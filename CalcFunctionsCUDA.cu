#include "hip/hip_runtime.h"
//Copyright (C) 2015, NRC "Kurchatov institute", http://www.nrcki.ru/e/engl.html, Moscow, Russia
//Author: Vladislav Neverov, vs-never@hotmail.com, neverov_vs@nrcki.ru
//
//This file is part of XaNSoNS.
//
//XaNSoNS is free software: you can redistribute it and / or modify
//it under the terms of the GNU General Public License as published by
//the Free Software Foundation, either version 3 of the License, or
//(at your option) any later version.
//
//XaNSoNS is distributed in the hope that it will be useful,
//but WITHOUT ANY WARRANTY; without even the implied warranty of
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
//GNU General Public License for more details.
//
//You should have received a copy of the GNU General Public License
//along with this program. If not, see <http://www.gnu.org/licenses/>.

//Contains host and device code for the CUDA version of XaNSoNS

#ifndef __HIPCC__
#define __HIPCC__
#endif
#include "typedefs.h"
#ifdef UseCUDA
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>

//Calculates rotational matrix (from CalcFunctions.cpp)
void calcRotMatrix(vect3d <double> *cf0, vect3d <double> *cf1, vect3d <double> *cf2, vect3d <double> euler, unsigned int convention);

//some float4 and float 3 functions (float4 used as float3)
inline __device__ __host__ float dot(float3 a, float3 b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
inline __device__ __host__ float dot(float3 a, float4 b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
inline __device__ __host__ float dot(float4 a, float3 b) { return a.x * b.x + a.y * b.y + a.z * b.z; }
inline __host__ __device__ float3 operator+(float3 a, float3 b){ return make_float3(a.x + b.x, a.y + b.y, a.z + b.z); }
inline __host__ __device__ float3 operator-(float3 a, float3 b){ return make_float3(a.x - b.x, a.y - b.y, a.z - b.z); }
inline __host__ __device__ float3 operator*(float3 a, float b){ return make_float3(a.x * b, a.y * b, a.z * b); }
inline __device__ float length(float3 v){ return sqrtf(dot(v, v)); }

//the following functions are used to calculate 2D diffraction patterns
//all the 2D arrays are flattened

/**
	Resets the 2D scattering intensity array

	@param *I   Intensity array
	@param Nq   Size of the scattering vector magnitude mesh (number of rows in the 2D intensity array)
	@param Nfi  Size of the scattering vector polar angle mesh (number of columns in the 2D intensity array)
*/
__global__ void zeroInt2DKernel(float *I, unsigned int Nq, unsigned int Nfi);

/**
	Resets the 2D scattering amplitude arrays (real and imaginary parts)

	@param *Ar  Real part of the 2D scattering amplitude array
	@param *Ai  Imaginary part of the 2D scattering amplitude array
	@param Nq   Size of the scattering vector magnitude mesh (number of rows in the 2D intensity array)
	@param Nfi  Size of the scattering vector polar angle mesh (number of columns in the 2D intensity array)
*/
__global__ void zeroAmp2DKernel(float *Ar, float *Ai, unsigned int Nq, unsigned int Nfi);

/**
	Computes the 2D scattering intensity using the scattering amplitude
	
	@param *I   Intensity array
	@param *Ar  Real part of the 2D scattering amplitude array
	@param *Ai  Imaginary part of the 2D scattering amplitude array
	@param Nq   Size of the scattering vector magnitude mesh (number of rows in the 2D amplitude array)
	@param Nfi  Size of the scattering vector polar angle mesh (number of columns in the 2D amplitude array)
*/
__global__ void Sum2DKernel(float *I, const float *Ar, const float *Ai, unsigned int Nq, unsigned int Nfi);

/**
	Multiplies the 2D scattering intensity by a normalizing factor

	@param *I   Intensity array
	@param Nq   Size of the scattering vector magnitude mesh (number of rows in the 2D intensity array)
	@param Nfi  Size of the scattering vector polar angle mesh (number of columns in the 2D intensity array)
	@param norm Normalizing factor
*/
__global__ void Norm2DKernel(float *I, unsigned int Nq, unsigned int Nfi, float norm);

/**
	Computes the polarization factor and multiplies the 2D scattering intensity by this factor

	@param *I     Intensity array
	@param Nq     Size of the scattering vector magnitude mesh (number of rows in the 2D intensity array)
	@param Nfi    Size of the scattering vector polar angle mesh (number of columns in the 2D intensity array)
	@param *q     Scattering vector magnitude array
	@param lambda Wavelength of the source
*/
template <unsigned int BlockSize2D> __global__ void PolarFactor2DKernel(float *I, unsigned int Nq, unsigned int Nfi, const float *q, float lambda);

/**
	Computes the real and imaginary parts of the 2D x-ray scattering amplitude in the polar coordinates (q,q_fi) of the reciprocal space

	@param *Ar    Real part of the 2D scattering amplitude array
	@param *Ai	  Imaginary part of the 2D scattering amplitude array
	@param *q     Scattering vector magnitude array
	@param Nq     Size of the scattering vector magnitude mesh (number of rows in the 2D amplitude array)
	@param Nfi    Size of the scattering vector polar angle mesh (number of columns in the 2D amplitude array)
	@param CS[]   Transposed rotational matrix. Defines the orientation of the nanoparticle in the 3D space.
	@param lambda Wavelength of the source
	@param *ra    Atomic coordinate array
	@param Nfin   Number of atoms to compute for in this kernel call (less or equal to the total number of atoms, cause the kernel is called iteratively in the loop)
	@param *FF    X-ray atomic form-factor array (for one kernel call the computations are done only for the atoms of the same chemical element)
*/
template <unsigned int BlockSize2D, unsigned int SizeR> __global__ void calcInt2DKernelXray(float *Ar, float *Ai, const float *q, unsigned int Nq, unsigned int Nfi, float3 CS[], float lambda, const float4 *ra, unsigned int Nfin, const float *FF);

/**
	Computes real and imaginary parts of the 2D neutron scattering amplitude in the polar coordinates (q,q_fi) of the reciprocal space

	@param *Ar    Real part of the 2D scattering amplitude array
	@param *Ai	  Imaginary part of the 2D scattering amplitude array
	@param *q     Scattering vector magnitude array
	@param Nq     Size of the scattering vector magnitude mesh (number of rows in the 2D amplitude array)
	@param Nfi    Size of the scattering vector polar angle mesh (number of columns in the 2D amplitude array)
	@param CS[]   Transposed rotational matrix. Defines the orientation of the nanoparticle in the 3D space.
	@param lambda Wavelength of the source
	@param *ra    Atomic coordinate array
	@param Nfin   Number of atoms to compute for in this kernel call (less or equal to the total number of atoms, cause the kernel is called iteratively in the loop)
	@param SL     Neutron scattering length of the current chemical element (for one kernel call the computations are done only for the atoms of the same chemical element)
*/
template <unsigned int BlockSize2D, unsigned int SizeR> __global__ void calcInt2DKernelNeutron(float *Ar, float *Ai, const float *q, unsigned int Nq, unsigned int Nfi, float3 CS[], float lambda, const float4 *ra, unsigned int Nfin, float SL);

/**
	Organazies the computations of the 2D scattering intensity in the polar coordinates (q,q_fi) of the reciprocal space with CUDA

	@param DeviceNUM  CUDA device number
	@param ***I2D     2D scattering intensity array (host). The memory is allocated inside the function.
	@param **I        1D (averaged over the polar angle) scattering intensity array (host). The memory is allocated inside the function.
	@param *cfg       Configuration of simulation parameters
	@param *NatomEl	  Array containing the total number of atoms of each chemical element (host)
	@param *ra        Atomic coordinate array (device)
	@param **dFF      X-ray atomic form-factor arrays for all chemical elements (device)
	@param SL         Vector with neutron scattering lengths for all chemical elements
	@param *dq        Scattering vector magnitude array (device)
*/
void calcInt2DCuda(int DeviceNUM, double ***I2D, double **I, const config *cfg, const unsigned int *NatomEl, const float4 *ra, const float * const *dFF, vector <double> SL, const float *dq);

//the following functions are used to calculate the histogram of interatomic distances

/**
	Resets the histogram array (unsigned long long int)

	@param *rij_hist  Histogram of interatomic distances
	@param N          Size of the array
*/
__global__ void zeroHistKernel(unsigned long long int *rij_hist, unsigned int N);

/**
	Computes the total histogram (first Nhist elements) using the partial histograms (for the devices with the CUDA compute capability < 2.0)

	@param *rij_hist   Partial histograms of interatomic distances
	@param Nhistcopies Number of the partial histograms to sum
	@param Nfin        Number of bins to compute for one kernel call
*/
__global__ void sumHistKernel(unsigned long long int *rij_hist, unsigned  int Nhistcopies, unsigned int Nfin, unsigned int Nhist);

/**
	Computes the histogram of interatomic distances

	@param *ri         Pointer to the coordinate of the 1st i-th atom in ra array
	@param *rj         Pointer to the coordinate of the 1st j-th atom in ra array
	@param iMax        Total number of i-th atoms for this kernel call
	@param jMax        Total number of j-th atoms for this kernel call
	@param *rij_hist   Histogram of interatomic distances
	@param bin         Width of the histogram bin
	@param Nhistcopies Number of partial histograms to compute (!=1 for the devices with the CUDA compute capability < 2.0 to reduce the number of atomicAdd() calls)
	@param Nhist       Size of the partial histogram of interatomic distances
	@param diag        True if the j-th atoms and the i-th atoms are the same (diagonal) for this kernel call
*/
template <unsigned int BlockSize2D> __global__ void calcHistKernel(const float4 *ri, const float4 *rj, unsigned int iMax, unsigned int jMax, unsigned long long int *rij_hist, float bin, unsigned int Nhistcopies, unsigned int Nhist, bool diag);

/**
	Organazies the computations of the histogram of interatomic distances with CUDA 

	@param DeviceNUM   CUDA device number
	@param **rij_hist  Histogram of interatomic distances (device). The memory is allocated inside the function.
	@param *ra         Atomic coordinate array (device)
	@param *NatomEl    Array containing the total number of atoms of each chemical element (host)
	@param Nel         Total number of different chemical elements in the nanoparticle
	@param Nhist       Size of the partial histogram of interatomic distances
	@param bin         Width of the histogram bin
*/
void calcHistCuda(int DeviceNUM, unsigned long long int **rij_hist, const float4 *ra, const unsigned int *NatomEl, unsigned int Nel, unsigned int Nhist, float bin);

//the following functions are used to calculate the powder diffraction pattern using the histogram of interatomic distances

/**
	Resets 1D float array of size N

	@param *A  Array
	@param N   Size of the array	
*/
__global__ void zero1DFloatArrayKernel(float *A, unsigned int N);

/**
	Computes the total scattering intensity (first Nq elements) from the partials sums computed by different thread blocks

	@param *I    Scattering intensity array
	@param Nq    Resolution of the total scattering intensity (powder diffraction pattern) 
	@param Nsum  Number of parts to sum (equalt to the total number of thread blocks in the grid)
*/
__global__ void sumIKernel(float *I, unsigned int Nq, unsigned int Nsum);

/**
	Adds the diagonal elements (j==i) of the Debye double sum to the x-ray scattering intensity 

	@param *I    Scattering intensity array
	@param *FF   X-ray atomic form-factor array (for one kernel call the computations are done only for the atoms of the same chemical element)
	@param Nq    Resolution of the total scattering intensity (powder diffraction pattern)
	@param N     Total number of atoms of the chemical element for whcich the computations are done 
*/
__global__ void addIKernelXray(float *I, const float *FF, unsigned int Nq, unsigned int N);

/**
	Adds the diagonal elements (j==i) of the Debye double sum to the neutron scattering intensity 

	@param *I    Scattering intensity array
	@param Nq    Resolution of the total scattering intensity (powder diffraction pattern)
	@param Add   The value to add to the intensity (the result of multiplying the square of the scattering length 
                 to the total number of atoms of the chemical element for whcich the computations are done) 
*/
__global__ void addIKernelNeutron(float *I, unsigned int Nq, float Add);

/**
	Computes polarization factor and multiplies scattering intensity by this factor

	@param *I     Scattering intensity array
	@param Nq     Size of the scattering intensity array
	@param *q     Scattering vector magnitude array
	@param lambda Wavelength of the source
*/
__global__ void PolarFactor1DKernel(float *I, unsigned int Nq, const float *q, float lambda);

/**
	Computes the x-ray scattering intensity (powder diffraction pattern) using the histogram of interatomic distances

	@param *I              Scattering intensity array
	@param *FFi            X-ray atomic form factor for the i-th atoms (all the i-th atoms are of the same chemical element for one kernel call)
	@param *FFj            X-ray atomic form factor for the j-th atoms (all the j-th atoms are of the same chemical element for one kernel call)
	@param *q              Scattering vector magnitude array
	@param Nq              Size of the scattering intensity array
	@param **rij_hist      Histogram of interatomic distances (device). The memory is allocated inside the function
	@param iBinSt          Starting index of the histogram bin for this kernel call (the kernel is called iteratively in a loop)
	@param Nhist           Size of the partial histogram of interatomic distances
	@param MaxBinsPerBlock Maximum number of histogram bins used by a single thread block
	@param bin             Width of the histogram bin
*/
template <unsigned int Size>__global__ void calcIntHistKernelXray(float *I, const float *FFi, const float *FFj, const float *q, unsigned int Nq, const unsigned long long int *rij_hist, unsigned int iBinSt, unsigned int Nhist, unsigned int MaxBinsPerBlock, float bin);

/**
	Computes the neutron scattering intensity (powder diffraction pattern) using the histogram of interatomic distances

	@param *I              Scattering intensity array
	@param SLij            Product of the scattering lenghts of i-th j-th atoms
	@param *q              Scattering vector magnitude array
	@param Nq              Size of the scattering intensity array
	@param **rij_hist      Histogram of interatomic distances (device). The memory is allocated inside the function
	@param iBinSt          Starting index of the histogram bin for this kernel call (the kernel is called iteratively in a loop)
	@param Nhist           Size of the partial histogram of interatomic distances
	@param MaxBinsPerBlock Maximum number of histogram bins used by a single thread block
	@param bin             Width of the histogram bin
*/
template <unsigned int Size>__global__ void calcIntHistKernelNeutron(float *I, float SLij, const float *q, unsigned int Nq, const unsigned long long int *rij_hist, unsigned int iBinSt, unsigned int Nhist, unsigned int MaxBinsPerBlock, float bin);

/**
	Organazies the computations of the scattering intensity (powder diffraction pattern) using the histogram of interatomic distances with CUDA

	@param DeviceNUM CUDA device number
	@param **I       Scattering intensity array (host). The memory is allocated inside the function
	@param *rij_hist Histogram of interatomic distances (device).
	@param *NatomEl  Array containing the total number of atoms of each chemical element (host)
	@param *cfg      Configuration of simulation parameters
	@param **dFF     X-ray atomic form-factor arrays for all chemical elements (device)
	@param SL        Vector with neutron scattering lengths for all chemical elements
	@param *dq       Scattering vector magnitude array (device)
	@param Ntot      Total number of atoms in the nanoparticle
*/
void calcInt1DHistCuda(int DeviceNUM, double **I, const unsigned long long int *rij_hist, const unsigned int *NatomEl, const config *cfg, const float * const * dFF, vector <double> SL, const float *dq, unsigned int Ntot);

//the following functions are used to calculate the PDFs

/**
	Computes the partial radial distribution function (RDF)

	@param *dPDF     Partial PDF array
	@param *rij_hist Histogram of interatomic distances (device)
	@param Nhist     Size of the partial histogram of interatomic distances
	@param mult      1 / (Ntot * bin_width)
*/
__global__ void calcPartialRDFkernel(float *dPDF, const unsigned long long int *rij_hist, unsigned int Nhist, float mult);

/**
	Computes the partial pair distribution function (PDF)

	@param *dPDF     Prtial PDF array
	@param *rij_hist Histogram of interatomic distances (device)
	@param Nhist     Size of the partial histogram of interatomic distances
	@param mult      1 / (4 * PI * rho * Ntot * bin_width)
	@param bin       Width of the histogram bin
*/
__global__ void calcPartialPDFkernel(float *dPDF, const unsigned long long int *rij_hist, unsigned int Nhist, float mult, float bin);

/**
	Computes the partial reduced pair distribution function (rPDF)

	@param *dPDF     Partial PDF array.
	@param *rij_hist Histogram of interatomic distances (device)
	@param Nhist     Size of the partial histogram of interatomic distances
	@param mult      1 / (Ntot * bin_width)
	@param submult   4 * PI * rho * NatomEl_i * NatomEl_j / SQR(Ntot)
	@param bin       Width of the histogram bin
*/
__global__ void calcPartialRPDFkernel(float *dPDF, const unsigned long long int *rij_hist, unsigned int Nhist, float mult, float submult, float bin);

/**
	Computes the total PDF using the partial PDFs

	@param *dPDF   Total (first Nhist elements) + partial PDF array. The memory is allocated inside the function.
	@param Nstart  Index of the first element of the partial PDF whcih will be added to the total PDF in this kernel call
	@param Nhist   Size of the partial histogram of interatomic distances
	@param multIJ  FF_i(q0) * FF_j(q0) / <FF> (for x-ray) and SL_i * SL_j / <SL> (for neutron)
*/
__global__ void calcPDFkernel(float *dPDF, unsigned int Nstart, unsigned int Nhist, float multIJ);

/**
	Depending on the computational scenario organazies the computations of the scattering intensity (powder diffraction pattern) or PDF using the histogram of interatomic distances with CUDA

	@param DeviceNUM CUDA device number
	@param **I       Scattering intensity array (host). The memory is allocated inside the function.
	@param **PDF     PDF array (host). The memory is allocated inside the function.
	@param *cfg      Configuration of simulation parameters
	@param *NatomEl  Array containing the total number of atoms of each chemical element (host)
	@param *ra       Atomic coordinate array (device)
	@param **dFF     X-ray atomic form-factor arrays for all chemical elements (device)
	@param SL        Vector with neutron scattering lengths for all chemical elements
	@param *dq       Scattering vector magnitude array (device)
*/
void calcPDFandDebyeCuda(int DeviceNUM, double **I, double **PDF, const config *cfg, const unsigned int *NatomEl, const float4 *ra, const float * const * dFF, vector<double> SL, const float *dq);

//the following functions are used to calculate the powder diffraction pattern using the original Debye equation (without the histogram approximation)

/**
	Computes the x-ray scattering intensity (powder diffraction pattern) using the histogram of interatomic distances

	@param *I    Scattering intensity array
	@param *FFi  X-ray atomic form factor for the i-th atoms (all the i-th atoms are of the same chemical element for one kernel call)
	@param *FFj  X-ray atomic form factor for the j-th atoms (all the j-th atoms are of the same chemical element for one kernel call)
	@param *q    Scattering vector magnitude array
	@param Nq    Size of the scattering intensity array
	@param *ri   Pointer to the coordinate of the 1st i-th atom in ra array
	@param *rj   Pointer to the coordinate of the 1st j-th atom in ra array
	@param iMax  Total number of i-th atoms for this kernel call
	@param jMax  Total number of j-th atoms for this kernel call
	@param diag  True if the j-th atoms and the i-th atoms are the same (diagonal) for this kernel call
*/
template <unsigned int BlockSize2D> __global__ void calcIntDebyeKernelXray(float *I, const float *FFi, const float *FFj, const float *q, unsigned int Nq, const float4 *ri, const float4 *rj, unsigned int iMax, unsigned int jMax, bool diag);

/**
	Computes the neutron scattering intensity (powder diffraction pattern) using the original Debye equation (without the histogram approximation)

	@param *I    Scattering intensity array
	@param SLij  Product of the scattering lenghts of i-th j-th atoms
	@param *q    Scattering vector magnitude array
	@param Nq    Size of the scattering intensity array
	@param *ri   Pointer to the coordinate of the 1st i-th atom in ra array
	@param *rj   Pointer to the coordinate of the 1st j-th atom in ra array
	@param iMax  Total number of i-th atoms for this kernel call
	@param jMax  Total number of j-th atoms for this kernel call
	@param diag  True if the j-th atoms and the i-th atoms are the same (diagonal) for this kernel call
*/
template <unsigned int BlockSize2D> __global__ void calcIntDebyeKernelNeutron(float *I, float SLij, const float *q, unsigned int Nq, const float4 *ri, const float4 *rj, unsigned int iMax, unsigned int jMax, bool diag);

/**
	Organazies the computations of the scattering intensity (powder diffraction pattern) using the original Debye equation (without the histogram approximation) with CUDA

	@param DeviceNUM CUDA device number
	@param **I       Scattering intensity array (host). The memory is allocated inside the function.
	@param *cfg      Configuration of simulation parameters
	@param *NatomEl  Array containing the total number of atoms of each chemical element (host)
	@param *ra       Atomic coordinate array (device)
	@param **dFF     X-ray atomic form-factor arrays for all chemical elements (device)
	@param SL        Vector with neutron scattering lengths for all chemical elements
	@param *dq       Scattering vector magnitude array (device)
*/
void calcIntDebyeCuda(int DeviceNUM, double **I, const config *cfg, const unsigned int *NatomEl, const float4 *ra, const float * const * dFF, vector<double> SL, const float *dq);

//the following functions are used to calculate the partial scattering intensities (for each pair of the structural blocks) using the original Debye equation (without the histogram approximation)

/**
	Computes the partial scattering intensity (*Ipart) from the partials sums (*I) computed by different thread blocks

	@param *I     Scattering intensity array (partials sums as computed by thread blocks)
	@param *Ipart Partial scattering intensity array
	@param Nq     Resolution of the total scattering intensity (powder diffraction pattern)
	@param Nsum   Number of parts to sum (equalt to the total number of thread blocks in the grid)
*/
__global__ void sumIpartialKernel(float *I, float *Ipart, unsigned int Nq, unsigned int Nsum);

/**
	Computes the total scattering intensity (powder diffraction pattern) using the partial scattering intensity

	@param *I     Partial + total (first Nq elements) scattering intensity array
	@param Nq     Resolution of the total scattering intensity (powder diffraction pattern)
	@param Npart  Number of the partial intensities to sum
*/
__global__ void integrateIpartialKernel(float *I, unsigned int Nq, unsigned int Nparts);

/**
	Organazies the computations of the scattering intensity (powder diffraction pattern) using the original Debye equation (without the histogram approximation) with CUDA

	@param DeviceNUM CUDA device number
	@param **I       Partial + total scattering intensity array (host). The memory is allocated inside the function.
	@param *cfg      Configuration of simulation parameters
	@param *NatomEl  Array containing the total number of atoms of each chemical element (host)
	@param *ra       Atomic coordinate array (device)
	@param **dFF     X-ray atomic form-factor arrays for all chemical elements (device)
	@param SL        Vector with neutron scattering lengths for all chemical elements
	@param *dq       Scattering vector magnitude array (device)
	@param *Block    Array of the structural blocks 
*/
void calcIntPartialDebyeCuda(int DeviceNUM, double **I, const config *cfg, const unsigned int *NatomEl, const float4 *ra, const float * const * dFF, vector<double> SL, const float *dq, const block *Block);

//the following functions are used to set the CUDA device, copy/delete the data to/from the device memory

/**
	Queries all CUDA devices. Checks and sets the CUDA device number
	Returns 0 if OK and -1 if no CUDA devices found

	@param *DeviceNUM CUDA device number
*/
int SetDeviceCuda(int *DeviceNUM);

/**
	Copies the atomic coordinates (ra), scattering vector magnitude (q) and the x-ray atomic form-factors (FF) to the device memory	

	@param *q      Scattering vector magnitude (host)
	@param *cfg    Configuration of simulation parameters
	@param *ra     Atomic coordinates (host)
	@param **dra   Atomic coordinates (device). The memory is allocated inside the function
	@param ***dFF  X-ray atomic form-factors (device). The memory is allocated inside the function
	@param **dq    Scattering vector magnitude (device). The memory is allocated inside the function
	@param FF      X-ray atomic form-factors (host)
	@param Ntot    Total number of atoms in the nanoparticle
*/
void dataCopyCUDA(const double *q, const config *cfg, const vector < vect3d <double> > *ra, float4 **dra, float ***dFF, float **dq, vector <double*> FF, unsigned int Ntot);

/**
	Deletes the atomic coordinates (ra), scattering vector magnitude (dq) and the x-ray atomic form-factors (dFF) from the device memory

	@param *ra    Atomic coordinates (device)
	@param **dFF  X-ray atomic form-factors (device)
	@param *dq    Scattering vector magnitude (device)
	@param Nel   Total number of different chemical elements in the nanoparticle
*/
void delDataFromDevice(float4 *ra, float **dFF, float *dq, unsigned int Nel);

/**
	Returns the theoretical peak performance of the CUDA device

	@param deviceProp  Device properties object
	@param show        If True, show the device information on screen
*/
unsigned int GetGFLOPS(hipDeviceProp_t deviceProp, bool show);

//Returns the theoretical peak performance of the CUDA device
unsigned int GetGFLOPS(hipDeviceProp_t deviceProp, bool show = false){
	unsigned int cc = deviceProp.major * 10 + deviceProp.minor; //compute capability
	unsigned int MP = deviceProp.multiProcessorCount; //number of multiprocessors
	unsigned int clockRate = deviceProp.clockRate / 1000; //GPU clockrate
	unsigned int GFLOPS = MP * 128 * 2 * clockRate / 1000; 
	switch (cc){
	case 10:
	case 11:
	case 12:
	case 13:
		GFLOPS = MP * 8 * 2 * clockRate / 1000;
		break;
	case 20:
		GFLOPS = MP * 32 * 2 * clockRate / 1000;
		break;
	case 21:
		GFLOPS = MP * 48 * 2 * clockRate / 1000;
		break;
	case 30:
	case 35:
	case 37:
		GFLOPS = MP * 192 * 2 * clockRate / 1000;
		break;
	case 50:
	case 52:
	case 61:
		GFLOPS = MP * 128 * 2 * clockRate / 1000;
		break;
	case 60:
		GFLOPS = MP * 64 * 2 * clockRate / 1000;
		break;	
	}
	if (show) {
		cout << "GPU name: " << deviceProp.name << "\n";
		cout << "CUDA compute capability: " << deviceProp.major << "." << deviceProp.minor << "\n";
		cout << "Number of multiprocessors: " << MP << "\n";
		cout << "GPU clock rate: " << clockRate << " MHz" << "\n";
		cout << "Theoretical peak performance: " << GFLOPS << " GFLOPs\n" << endl;
	}
	return GFLOPS;
}

//Resets the 2D scattering intensity array
__global__ void zeroInt2DKernel(float *I, unsigned int Nq, unsigned int Nfi){
	unsigned int iq = blockDim.y * blockIdx.y + threadIdx.y, ifi = blockDim.x * blockIdx.x + threadIdx.x;
	if ((iq < Nq) && (ifi < Nfi))	I[iq*Nfi + ifi] = 0;
}

//Resets the 2D scattering amplitude arrays (real and imaginary parts)
__global__ void zeroAmp2DKernel(float *Ar, float *Ai, unsigned int Nq, unsigned int Nfi){
	unsigned int iq = blockDim.y * blockIdx.y + threadIdx.y, ifi = blockDim.x * blockIdx.x + threadIdx.x;
	if ((iq < Nq) && (ifi < Nfi)){
		Ar[iq*Nfi + ifi] = 0;
		Ai[iq*Nfi + ifi] = 0;
	}
}

//Computes the 2D scattering intensity using the scattering amplitude
__global__ void Sum2DKernel(float *I,const float *Ar,const float *Ai, unsigned int Nq, unsigned int Nfi){
	unsigned int iq = blockDim.y * blockIdx.y + threadIdx.y, ifi = blockDim.x * blockIdx.x + threadIdx.x;
	if ((iq < Nq) && (ifi < Nfi))	I[iq * Nfi + ifi] += SQR(Ar[iq * Nfi + ifi]) + SQR(Ai[iq * Nfi + ifi]);
}

//Multiplies the 2D scattering intensity by a normalizing factor
__global__ void Norm2DKernel(float *I, unsigned int Nq, unsigned int Nfi, float norm){
	unsigned int iq = blockDim.y * blockIdx.y + threadIdx.y, ifi = blockDim.x * blockIdx.x + threadIdx.x;
	if ((iq < Nq) && (ifi < Nfi))	I[iq * Nfi + ifi] *= norm;
}

//Computes the polarization factor and multiplies the 2D scattering intensity by this factor
template <unsigned int BlockSize2D> __global__ void PolarFactor2DKernel(float *I, unsigned int Nq, unsigned int Nfi, const float *q, float lambda){
	unsigned int iq = BlockSize2D * blockIdx.y + threadIdx.y, ifi = BlockSize2D * blockIdx.x + threadIdx.x;
	unsigned int iqCopy = BlockSize2D * blockIdx.y + threadIdx.x;
	__shared__ float factor[BlockSize2D];
	if ((threadIdx.y == 0) && (iqCopy < Nq)) {
		//polarization factor is computed only by the threads of the first warp (half-warp for the devices with CC < 2.0) and stored in the shared memory
		float sintheta = q[iqCopy] * (lambda * 0.25f / PIf);
		float cos2theta = 1.f - 2.f * SQR(sintheta);
		factor[threadIdx.x] = 0.5f * (1.f + SQR(cos2theta));
	}
	__syncthreads();
	if ((iq < Nq) && (ifi < Nfi)) I[iq * Nfi + ifi] *= factor[threadIdx.y]; 
}

//Computes polarization factor and multiplies scattering intensity by this factor
__global__ void PolarFactor1DKernel(float *I, unsigned int Nq, const float *q, float lambda){
	unsigned int iq = blockIdx.x * blockDim.x + threadIdx.x;
	if (iq < Nq)	{
		float sintheta = q[iq] * (lambda * 0.25f / PIf);
		float cos2theta = 1.f - 2.f * SQR(sintheta);
		float factor = 0.5f * (1.f + SQR(cos2theta));
		I[blockIdx.y * Nq + iq] *= factor;
	}
}

//Computes the real and imaginary parts of the 2D x - ray scattering amplitude in the polar coordinates(q, q_fi) of the reciprocal space
template <unsigned int BlockSize2D, unsigned int SizeR> __global__ void calcInt2DKernelXray(float *Ar, float *Ai, const float *q, unsigned int Nq, unsigned int Nfi, float3 CS[], float lambda, const float4 *ra, unsigned int Nfin, const float *FF){
	//to avoid bank conflicts for shared memory operations BlockSize2D should be equal to the size of the warp (or half-warp for the devices with the CC < 2.0)
	//SizeR should be a multiple of BlockSize2D
	unsigned int iq = BlockSize2D * blockIdx.y + threadIdx.y, ifi = BlockSize2D * blockIdx.x + threadIdx.x; //each thread computes only one element of the 2D amplitude matrix
	unsigned int iqCopy = BlockSize2D * blockIdx.y + threadIdx.x;//copying of the scattering vector magnitude to the shared memory is performed by the threads of the same warp (half-warp)
	__shared__ float lFF[BlockSize2D]; //cache array for the x-ray  atomic from-factors
	__shared__ float qi[BlockSize2D]; //cache array for the scattering vector magnitude
	__shared__ float4 r[SizeR]; //cache array for the atomic coordinates
	unsigned int Niter = Nfin / SizeR + BOOL(Nfin % SizeR);//we don't have enough shared memory to load the array of atomic coordinates as a whole, so we do it with iterations
	float3 qv; //scattering vector
	float lAr = 0, lAi = 0, cosfi = 0, sinfi = 0, sintheta = 0, costheta = 0;
	if ((threadIdx.y == 0) && (iqCopy < Nq)) lFF[threadIdx.x] = FF[iqCopy]; //loading x-ray atomic form-factors to the shared memory (only threads from the first warp (half-warp) are used)
	if ((threadIdx.y == 2) && (iqCopy < Nq)) qi[threadIdx.x] = q[iqCopy]; //loading scattering vector magnitude to the shared memory (only threads from the third warp (first half of the second warp) are used)
	__syncthreads(); //synchronizing after loading to the shared memory
	if ((iq < Nq) && (ifi < Nfi)){//checking the 2d array margins
		__sincosf(ifi * 2.f * PIf / Nfi, &sinfi, &cosfi); //computing sin(fi), cos(fi)
		sintheta = 0.25f * lambda * qi[threadIdx.y] / PIf; //q = 4pi/lambda*sin(theta)
		costheta = 1.f - SQR(sintheta); //theta in [0, pi/2];
		qv = make_float3(costheta * cosfi, costheta * sinfi, -sintheta) * qi[threadIdx.y]; //computing the scattering vector
		//instead of pre-multiplying the atomic coordinates by the rotational matrix we are pre-multiplying the scattering vector by the transposed rotational matrix (dot(qv,r) will be the same)
		qv = make_float3(dot(qv, CS[0]), dot(qv, CS[1]), dot(qv, CS[2]));
	}
	for (unsigned int iter = 0; iter < Niter; iter++){
		unsigned int NiterFin = MIN(Nfin - iter * SizeR, SizeR); //checking for the margins of the atomic coordinates array
		if (threadIdx.y < SizeR / BlockSize2D) {
			unsigned int iAtom = threadIdx.y * BlockSize2D + threadIdx.x; 
			if (iAtom < NiterFin) r[iAtom] = ra[iter * SizeR + iAtom]; //loading the atomic coordinates to the shared memory
		}
		__syncthreads(); //synchronizing after loading to shared memory
		if ((iq < Nq) && (ifi < Nfi)){//checking the 2d array margins
			for (unsigned int iAtom = 0; iAtom < NiterFin; iAtom++){
				__sincosf(dot(qv, r[iAtom]), &sinfi, &cosfi); //cos(dot(qv*r)), sin(dot(qv,r))
				lAr += cosfi; //real part of the amplitute
				lAi += sinfi; //imaginary part of the amplitute
			}
		}
		__syncthreads(); //synchronizing before the next loading starts
	}
	if ((iq < Nq) && (ifi < Nfi)){//checking the 2d array margins
		Ar[iq * Nfi + ifi] += lFF[threadIdx.y] * lAr; //multiplying the real part of the amplitude by the form-factor and writing the results to the global memory
		Ai[iq * Nfi + ifi] += lFF[threadIdx.y] * lAi; //doing the same for the imaginary part of the amplitude
	}	
}

//Computes real and imaginary parts of the 2D neutron scattering amplitude in the polar coordinates (q,q_fi) of the reciprocal space
template <unsigned int BlockSize2D, unsigned int SizeR> __global__ void calcInt2DKernelNeutron(float *Ar, float *Ai, const float *q, unsigned int Nq, unsigned int Nfi, float3 CS[], float lambda, const float4 *ra, unsigned int Nfin, float SL){
	//see comments in the calcInt2DKernelXray() kernel
	unsigned int iq = BlockSize2D * blockIdx.y + threadIdx.y, ifi = BlockSize2D * blockIdx.x + threadIdx.x; 
	unsigned int iqCopy = BlockSize2D * blockIdx.y + threadIdx.x;
	__shared__ float qi[BlockSize2D]; 
	__shared__ float4 r[SizeR];	
	unsigned int Niter = Nfin / SizeR + BOOL(Nfin % SizeR);
	float3 qv;
	float lAr = 0, lAi = 0, cosfi = 0, sinfi = 0, sintheta = 0, costheta = 0;
	if ((threadIdx.y == 0) && (iqCopy < Nq)) qi[threadIdx.x] = q[iqCopy];
	__syncthreads();
	if ((iq < Nq) && (ifi < Nfi)){
		__sincosf(ifi * 2.f * PIf / Nfi, &sinfi, &cosfi);
		sintheta = 0.25f * lambda*qi[threadIdx.y] / PIf;
		costheta = 1.f - SQR(sintheta);
		qv = make_float3(costheta*cosfi, costheta * sinfi, -sintheta) * qi[threadIdx.y];
		qv = make_float3(dot(qv, CS[0]), dot(qv, CS[1]), dot(qv, CS[2]));
	}
	for (unsigned int iter = 0; iter < Niter; iter++){
		unsigned int NiterFin = MIN(Nfin - iter * SizeR, SizeR);
		if (threadIdx.y < SizeR / BlockSize2D) {
			unsigned int iAtom = threadIdx.y * BlockSize2D + threadIdx.x; 
			if (iAtom < NiterFin) r[iAtom] = ra[iter * SizeR + iAtom];
		}
		__syncthreads();
		if ((iq < Nq) && (ifi < Nfi)){
			for (unsigned int iAtom = 0; iAtom < NiterFin; iAtom++){
				__sincosf(dot(qv, r[iAtom]), &sinfi, &cosfi);
				lAr += cosfi;
				lAi += sinfi;
			}
		}
		__syncthreads();
	}
	if ((iq < Nq) && (ifi < Nfi)){
		Ar[iq * Nfi + ifi] += SL * lAr;
		Ai[iq * Nfi + ifi] += SL * lAi;
	}
}

//Organazies the computations of the 2D scattering intensity in the polar coordinates(q, q_fi) of the reciprocal space with CUDA
void calcInt2DCuda(int DeviceNUM, double ***I2D, double **I, const config *cfg, const unsigned int *NatomEl, const float4 *ra, const float * const *dFF, vector<double> SL, const float *dq){
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	unsigned int MaxAtomsPerLaunch = 0, BlockSize2D = BlockSize2Dsmall, Ntot = 0;
	float *hI, *dI, *dAr, *dAi;
	*I = new double[cfg->q.N]; //array for 1d scattering intensity I[q] (I2D[q][fi] averaged over polar angle fi)
	*I2D = new double*[cfg->q.N]; //array for 2d scattering intensity 
	for (unsigned int iq = 0; iq < cfg->q.N; iq++){
		(*I)[iq] = 0;
		(*I2D)[iq] = new double[cfg->Nfi];
	}
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DeviceNUM); //getting device information
	unsigned int GFLOPS = GetGFLOPS(deviceProp); //theoretical peak GPU performance
	if (deviceProp.kernelExecTimeoutEnabled){ //killswitch is enabled, so the time limit should not be exceeded
		const double tmax = 0.02; //maximum kernel execution time in seconds
		const double k = 4.e-8; // t = k * MaxAtomsPerLaunch * Nq * Nfi / GFLOPS
		MaxAtomsPerLaunch = (unsigned int)((tmax * GFLOPS) / (k * cfg->q.N * cfg->Nfi)); //maximum number of atoms per kernel launch
	}
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) Ntot += NatomEl[iEl]; //total number of atoms
	unsigned int Nm = cfg->q.N * cfg->Nfi; //dimension of 2D intensity array
	hI = new float[Nm]; //host array for 2D intensity
	//allocating memory on the device for amplitude and intensity 2D arrays
	//GPU has linear memory, so we stretch 2D arrays into 1D arrays
	hipMalloc(&dAr, Nm * sizeof(float));
	hipMalloc(&dAi, Nm * sizeof(float));
	hipMalloc(&dI, Nm * sizeof(float));	
	hipDeviceSynchronize(); //synchronizing before calculating the amplitude
	dim3 dimBlock(BlockSize2D, BlockSize2D); //2d thread block size
	dim3 dimGrid(cfg->Nfi / BlockSize2D + BOOL(cfg->Nfi % BlockSize2D), cfg->q.N / BlockSize2D + BOOL(cfg->q.N % BlockSize2D)); //grid size
	float3 CS[3], *dCS; //three rows of the transposed rotational matrix for the host and the device
	unsigned int Nst, Nfin;
	//2d scattering intensity should be calculated for the preset orientation of the sample (or averaged over multiple orientations specified by mesh)
	double dalpha = (cfg->Euler.max.x - cfg->Euler.min.x) / cfg->Euler.N.x, dbeta = (cfg->Euler.max.y - cfg->Euler.min.y) / cfg->Euler.N.y, dgamma = (cfg->Euler.max.z - cfg->Euler.min.z) / cfg->Euler.N.z;
	if (cfg->Euler.N.x < 2) dalpha = 0;
	if (cfg->Euler.N.y < 2) dbeta = 0;
	if (cfg->Euler.N.z < 2) dgamma = 0;
	hipMalloc(&dCS, 3 * sizeof(float3)); //allocating the device memory for the transposed rotational matrix
	zeroInt2DKernel << <dimGrid, dimBlock >> >(dI, cfg->q.N, cfg->Nfi); //reseting the 2D intensity matrix
	vect3d <double> cf0, cf1, cf2; //three rows of the rotational matrix
	for (unsigned int ia = 0; ia < cfg->Euler.N.x; ia++){
		double alpha = cfg->Euler.min.x + (ia + 0.5)*dalpha;
		for (unsigned int ib = 0; ib < cfg->Euler.N.y; ib++){
			double beta = cfg->Euler.min.y + (ib + 0.5)*dbeta;
			for (unsigned int ig = 0; ig < cfg->Euler.N.z; ig++){
				double gamma = cfg->Euler.min.z + (ig + 0.5)*dgamma;
				vect3d <double> euler(alpha, beta, gamma);
				calcRotMatrix(&cf0, &cf1, &cf2, euler, cfg->EulerConvention); //calculating the rotational matrix
				CS[0]=make_float3(float(cf0.x), float(cf1.x), float(cf2.x)); //transposing the rotational matrix
				CS[1]=make_float3(float(cf0.y), float(cf1.y), float(cf2.y));
				CS[2]=make_float3(float(cf0.z), float(cf1.z), float(cf2.z));
				hipMemcpy(dCS, CS, 3 * sizeof(float3), hipMemcpyHostToDevice); //copying transposed rotational matrix from the host memory to the device memory 
				zeroAmp2DKernel << <dimGrid, dimBlock >> >(dAr, dAi, cfg->q.N, cfg->Nfi); //reseting 2D amplitude arrays
				hipDeviceSynchronize(); //synchronizing before calculation starts to ensure that amplitude arrays were successfully set to zero
				unsigned int inp = 0;
				for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++){ //looping over chemical elements (or ions)
					if (MaxAtomsPerLaunch) { //killswitch is enabled so MaxAtomsPerLaunch is set
						for (unsigned int i = 0; i < NatomEl[iEl] / MaxAtomsPerLaunch + BOOL(NatomEl[iEl] % MaxAtomsPerLaunch); i++) { //looping over the iterations
							Nst = inp + i*MaxAtomsPerLaunch; //index for the first atom on the current iteration step
							Nfin = MIN(Nst + MaxAtomsPerLaunch, inp + NatomEl[iEl]) - Nst; //index for the last atom on the current iteration step
							//float time; //time control sequence
							//hipEvent_t start, stop;
							//hipEventCreate(&start);
							//hipEventCreate(&stop);
							//hipEventRecord(start, 0);
							if (cfg->source == xray) {
								calcInt2DKernelXray <BlockSize2Dsmall, 8 * BlockSize2Dsmall> << <dimGrid, dimBlock >> >(dAr, dAi, dq, cfg->q.N, cfg->Nfi, dCS, float(cfg->lambda), ra + Nst, Nfin, dFF[iEl]);
							}
							else {//neutron scattering
								calcInt2DKernelNeutron <BlockSize2Dsmall, 8 * BlockSize2Dsmall> << <dimGrid, dimBlock >> >(dAr, dAi, dq, cfg->q.N, cfg->Nfi, dCS, float(cfg->lambda), ra + Nst, Nfin, float(SL[iEl]));
							}
							hipDeviceSynchronize(); //synchronizing to ensure that additive operations does not overlap
							//hipEventRecord(stop, 0);
							//hipEventSynchronize(stop);
							//hipEventElapsedTime(&time, start, stop);
							//cout << "calcInt2DKernel execution time is: " << time << " ms\n" << endl;
						}
					}
					else { //killswitch is disabled so we execute the kernels for the entire ensemble of atoms
						Nst = inp;
						Nfin = NatomEl[iEl];
						if (cfg->source == xray) {
							calcInt2DKernelXray <BlockSize2Dsmall, 8 * BlockSize2Dsmall> << <dimGrid, dimBlock >> >(dAr, dAi, dq, cfg->q.N, cfg->Nfi, dCS, float(cfg->lambda), ra + Nst, Nfin, dFF[iEl]);
						}
						else {//neutron scattering
							calcInt2DKernelNeutron <BlockSize2Dsmall, 8 * BlockSize2Dsmall> << <dimGrid, dimBlock >> >(dAr, dAi, dq, cfg->q.N, cfg->Nfi, dCS, float(cfg->lambda), ra + Nst, Nfin, float(SL[iEl]));
						}
						hipDeviceSynchronize(); //synchronizing to ensure that additive operations does not overlap
					}
					inp += NatomEl[iEl];
				}				
				Sum2DKernel << <dimGrid, dimBlock >> >(dI, dAr, dAi, cfg->q.N, cfg->Nfi); //calculating the 2d scattering intensity by the scattering amplitude
			}
		}
	}
	float norm = 1.f / (Ntot*cfg->Euler.N.x*cfg->Euler.N.y*cfg->Euler.N.z); //normalizing factor
	Norm2DKernel << <dimGrid, dimBlock >> >(dI, cfg->q.N, cfg->Nfi, norm); //normalizing the 2d scattering intensity
	hipDeviceSynchronize(); //synchronizing to ensure that multiplying operations does not overlap
	if (cfg->PolarFactor) { //multiplying the 2d intensity by polar factor
		PolarFactor2DKernel <BlockSize2Dsmall> << <dimGrid, dimBlock >> >(dI, cfg->q.N, cfg->Nfi, dq, float(cfg->lambda));
	}
	hipMemcpy(hI, dI, Nm*sizeof(float), hipMemcpyDeviceToHost);  //copying the 2d intensity matrix from the device memory to the host memory 
	for (unsigned int iq = 0; iq < cfg->q.N; iq++){
		for (unsigned int ifi = 0; ifi < cfg->Nfi; ifi++)	{
			(*I2D)[iq][ifi] = double(hI[iq * cfg->Nfi + ifi]);
			(*I)[iq] += (*I2D)[iq][ifi]; //calculating the 1d intensity (averaging I2D[q][fi] over the polar angle fi)
		}
		(*I)[iq] /= cfg->Nfi;
	}
	//deallocating the device memory
	hipFree(dCS);
	hipFree(dAr);
	hipFree(dAi);
	hipFree(dI);
	//deallocating the host memory
	delete[] hI;
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "2D pattern calculation time: " << time/1000 << " s" << endl;
}

//Resets 1D float array of size N
__global__ void zero1DFloatArrayKernel(float *A, unsigned int N){
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i<N) A[i]=0;
}

//Adds the diagonal elements(j == i) of the Debye double sum to the x - ray scattering intensity
__global__ void addIKernelXray(float *I, const float *FF, unsigned int Nq, unsigned int N) {
	unsigned int iq = blockIdx.x * blockDim.x + threadIdx.x;
	if (iq < Nq)	{
		float lFF = FF[iq];
		I[iq] += SQR(lFF) * N;
	}
}

//Adds the diagonal elements(j == i) of the Debye double sum to the neutron scattering intensity
__global__ void addIKernelNeutron(float *I, unsigned int Nq, float Add) {
	unsigned int iq = blockIdx.x * blockDim.x + threadIdx.x;
	if (iq < Nq)	I[iq] += Add;
}

//Computes the total scattering intensity (first Nq elements) from the partials sums computed by different thread blocks
__global__ void sumIKernel(float *I, unsigned int Nq, unsigned int Nsum){
	unsigned int iq = blockDim.x * blockIdx.x + threadIdx.x;
	if (iq<Nq) {
		for (unsigned int j = 1; j < Nsum; j++)	I[iq] += I[j * Nq + iq];
	}
}

//Resets the histogram array (unsigned long long int)
__global__ void zeroHistKernel(unsigned long long int *rij_hist,unsigned int N){
	unsigned int i=blockDim.x * blockIdx.x + threadIdx.x;
	if (i<N) rij_hist[i]=0;
}	

//Computes the histogram of interatomic distances
template <unsigned int BlockSize2D> __global__ void calcHistKernel(const float4 *ri,const float4 *rj, unsigned int iMax, unsigned int jMax, unsigned long long int *rij_hist, float bin, unsigned int Nhistcopies, unsigned int Nhist, bool diag){
	if ((diag) && (blockIdx.x < blockIdx.y)) return; //we need to calculate inter-atomic distances only for j > i, so if we are in the diagonal grid, all the subdiagonal blocks (for which j < i for all threads) do nothing and return
	unsigned int jt = threadIdx.x, it = threadIdx.y;
	unsigned int j = blockIdx.x * BlockSize2D + jt;
	unsigned int iCopy = blockIdx.y * BlockSize2D + jt; //jt!!! memory transaction are performed by the threads of the same warp to coalesce them
	unsigned int i = blockIdx.y * BlockSize2D + it;
	unsigned int copyind = 0;
	if (Nhistcopies>1) copyind = ((it * BlockSize2D + jt) % Nhistcopies) * Nhist; //some optimization for CC < 2.0. Making multiple copies of the histogram array reduces the number of atomicAdd() operations on the same elements.
	__shared__ float4 ris[BlockSize2D], rjs[BlockSize2D]; //cache arrays for atomic coordinates (we use float3 here to avoid bank conflicts)
	if ((it == 0) && (j < jMax)) { //copying atomic coordinates for j-th (column) atoms (only the threads of the first half-warp are used)
		rjs[jt] = rj[j];
	}
	if ((it == 2) && (iCopy < iMax)) { //the same for i-th (row) atoms (only the threads of the first half-warp of the second warp for CC < 2.0 are used)
		ris[jt] = ri[iCopy];
	}
	__syncthreads(); //sync to ensure that copying is complete
	if (!diag){
		if ((j < jMax) && (i < iMax)) {
			float rij = sqrtf(SQR(ris[it].x - rjs[jt].x) + SQR(ris[it].y - rjs[jt].y) + SQR(ris[it].z - rjs[jt].z));//calculate distance
			unsigned int index = (unsigned int)(rij / bin); //get the index of histogram bin
			atomicAdd(&rij_hist[copyind + index], 1); //add +1 to histogram bin
		}
	}
	else{//we are in diagonal grid
		if ((j < jMax) && (i < iMax) && (j > i)) {//all the subdiagonal blocks already quit, but we have diagonal blocks  (blockIdx.x == blockIdx.y), so we should check if j > i
			float rij = sqrtf(SQR(ris[it].x - rjs[jt].x) + SQR(ris[it].y - rjs[jt].y) + SQR(ris[it].z - rjs[jt].z));
			unsigned int index = (unsigned int)(rij / bin);
			atomicAdd(&rij_hist[copyind + index], 1);
		}
	}
}

//Computes the total histogram (first Nhist elements) using the partial histograms (for the devices with the CUDA compute capability < 2.0)
__global__ void sumHistKernel(unsigned long long int *rij_hist, unsigned  int Nhistcopies, unsigned int Nfin, unsigned int Nhist){
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < Nfin){
		for (unsigned int iCopy = 1; iCopy < Nhistcopies; iCopy++)	rij_hist[i] += rij_hist[Nhist * iCopy + i];
	}
}

//Organazies the computations of the histogram of interatomic distances with CUDA 
void calcHistCuda(int DeviceNUM, unsigned long long int **rij_hist, const float4 *ra, const unsigned int *NatomEl, unsigned int Nel, unsigned int Nhist, float bin){
	unsigned int GridSizeExecMax = 2048;
	unsigned int BlockSize = BlockSize1Dsmall, BlockSize2D = BlockSize2Dsmall; //size of the thread blocks (256, 16x16)
	unsigned int Nhistcopies = 1, NhistEl = (Nel * (Nel + 1)) / 2 * Nhist;//NhistEl - number of partial (Element1<-->Element2) histograms
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DeviceNUM); //getting the device properties
	int cc = deviceProp.major * 10 + deviceProp.minor; //device compute capability
	if (cc<20){//optimization for the devices with CC < 2.0
		//atomic operations work very slow for the devices with Tesla architecture as compared with the modern devices
		//we minimize the number of atomic operations on the same elements by making multiple copies of pair-distribution histograms
		size_t free, total;
		hipMemGetInfo(&free, &total); //checking the amount of the free GPU memory	
		Nhistcopies = MIN(BlockSize,(unsigned int)(0.25 * float(free) / (NhistEl * sizeof(unsigned long long int)))); //set optimal number for histogram copies 
		if (!Nhistcopies) Nhistcopies = 1;
	}
	unsigned int GFLOPS = GetGFLOPS(deviceProp); //theoretical peak GPU performance
	if (deviceProp.kernelExecTimeoutEnabled)	{//killswitch is enabled, so the time limit should not be exceeded
		const double tmax = 0.02; //maximum kernel time execution in seconds
		const double k = 1.e-6; // t = k * GridSizeExecMax^2 * BlockSize2D^2 / GFLOPS
		GridSizeExecMax = MIN((unsigned int)(sqrt(tmax * GFLOPS / k) / BlockSize2D), GridSizeExecMax);
	}
	//total histogram size is equal to the product of: partial histogram size for one pair of elements (Nhist), number of partial histograms ((Nel*(Nel + 1)) / 2), number of histogram copies (Nhistcopies)
	unsigned int NhistTotal = NhistEl * Nhistcopies;
	hipError_t err = hipMalloc(rij_hist, NhistTotal * sizeof(unsigned long long int));//trying to allocate large amount of memory, check for errors
	if (err != hipSuccess) cout << "Error in calcHistCuda(), hipMalloc(): " << hipGetErrorString(err) << endl;
	unsigned int GSzero = MIN(65535, NhistTotal / BlockSize + BOOL(NhistTotal % BlockSize));//Size of the grid for zeroHistKernel (it could not be large than 65535)
	//reseting pair-distribution histogram array
	for (unsigned int iter = 0; iter < NhistTotal / BlockSize + BOOL(NhistTotal % BlockSize); iter += GSzero)	zeroHistKernel << < GSzero, BlockSize >> >(*rij_hist + iter*BlockSize, NhistTotal - iter*BlockSize);
	hipDeviceSynchronize();//synchronizing before the calculation starts
	dim3 blockgrid(BlockSize2D, BlockSize2D);//2D thread block size
	unsigned int Nstart = 0, jAtom0, iAtomST = 0;
	bool diag = false;
	for (unsigned int iEl = 0; iEl < Nel; iAtomST += NatomEl[iEl], iEl++) {
		unsigned int jAtomST = iAtomST;
		for (unsigned int jEl = iEl; jEl < Nel; jAtomST += NatomEl[jEl], jEl++, Nstart += Nhist) {//each time we move to the next pair of elements (iEl,jEl) we also move to the respective part of histogram (Nstart += Nhist)
			for (unsigned int iAtom = 0; iAtom < NatomEl[iEl]; iAtom += BlockSize2D * GridSizeExecMax){
				unsigned int GridSizeExecY = MIN((NatomEl[iEl] - iAtom) / BlockSize2D + BOOL((NatomEl[iEl] - iAtom) % BlockSize2D), GridSizeExecMax);//Y-size of the grid on the current step
				unsigned int iMax = MIN(BlockSize2D * GridSizeExecY, NatomEl[iEl] - iAtom);//index of the last i-th (row) atom
				(iEl == jEl) ? jAtom0 = iAtom : jAtom0 = 0;//loop should exclude subdiagonal grids
				for (unsigned int jAtom = jAtom0; jAtom < NatomEl[jEl]; jAtom += BlockSize2D * GridSizeExecMax){
					unsigned int GridSizeExecX = MIN((NatomEl[jEl] - jAtom) / BlockSize2D + BOOL((NatomEl[jEl] - jAtom) % BlockSize2D), GridSizeExecMax);//X-size of the grid on the current step
					unsigned int jMax = MIN(BlockSize2D * GridSizeExecX, NatomEl[jEl] - jAtom);//index of the last j-th (column) atom
					dim3 grid(GridSizeExecX, GridSizeExecY);
					(iAtomST + iAtom == jAtomST + jAtom) ? diag = true : diag = false;//checking if we are on the diagonal grid or not
					/*float time;
					hipEvent_t start, stop;
					hipEventCreate(&start);
					hipEventCreate(&stop);
					hipEventRecord(start, 0);*/
					calcHistKernel <BlockSize2Dsmall> << <grid, blockgrid >> >(ra + iAtomST + iAtom, ra + jAtomST + jAtom, iMax, jMax, *rij_hist + Nstart, bin, Nhistcopies, NhistEl, diag);
					if (deviceProp.kernelExecTimeoutEnabled) hipDeviceSynchronize();//the kernel above uses atomic operation, it's hard to predict the execution time of a single kernel, so sync to avoid the killswitch triggering 
					/*hipEventRecord(stop, 0);
					hipEventSynchronize(stop);
					hipEventElapsedTime(&time, start, stop);
					cout << "calcHistKernel execution time is: " << time << " ms\n" << endl;*/
				}
			}
		}
	}
	hipDeviceSynchronize();//synchronizing to ensure that all calculations ended before histogram copies summation starts
	if (Nhistcopies>1) {//summing the histogram copies
		unsigned int GSsum = MIN(65535, NhistEl / BlockSize + BOOL(NhistEl % BlockSize));
		for (unsigned int iter = 0; iter < NhistEl / BlockSize + BOOL(NhistEl % BlockSize); iter += GSsum)	sumHistKernel << <GSsum, BlockSize >> >(*rij_hist + iter * BlockSize, Nhistcopies, NhistEl - iter * BlockSize, NhistEl);
	}
	hipDeviceSynchronize();//synchronizing before the further usage of histogram in other functions
}

//Computes the x-ray scattering intensity (powder diffraction pattern) using the histogram of interatomic distances
template <unsigned int Size>__global__ void calcIntHistKernelXray(float *I, const float *FFi, const float *FFj, const float *q, unsigned int Nq, const unsigned long long int *rij_hist, unsigned int iBinSt, unsigned int Nhist, unsigned int MaxBinsPerBlock, float bin){
	__shared__ long long int Nrij[Size];//cache array for the histogram
	Nrij[threadIdx.x] = 0;
	__syncthreads();
	unsigned int iBegin = iBinSt + blockIdx.x * MaxBinsPerBlock;//first index for histogram bin to process
	unsigned int iEnd = MIN(Nhist, iBegin + MaxBinsPerBlock);//last index for histogram bin to process
	if (iEnd < iBegin) return;
	unsigned int Niter = (iEnd - iBegin) / blockDim.x + BOOL((iEnd - iBegin) % blockDim.x);//number of iterations
	for (unsigned int iter = 0; iter < Niter; iter++){//we don't have enough shared memory to load the histogram array as a whole, so we do it with iterations
		unsigned int NiterFin = MIN(iEnd - iBegin - iter * blockDim.x, blockDim.x);//maximum number of histogram bins on current iteration step
		if (threadIdx.x < NiterFin) Nrij[threadIdx.x] = rij_hist[iBegin + iter * blockDim.x + threadIdx.x];//loading the histogram array to the shared memory
		__syncthreads();//synchronizing after loading
		for (unsigned int iterq = 0; iterq < (Nq / blockDim.x) + BOOL(Nq % blockDim.x); iterq++) {//if Nq > blockDim.x there will be threads that compute more than one element of the intensity array
			unsigned int iq = iterq*blockDim.x + threadIdx.x;//index of the intensity array element
			if (iq < Nq) {//checking for the array margin
				float lI=0, qrij;
				float lq = q[iq];//copying the scattering vector magnitude to the local memory
				for (unsigned int i = 0; i < NiterFin; i++) {//looping over the histogram bins
					if (Nrij[i]){
						qrij = lq * ((float)(iBegin + iter * blockDim.x + i) + 0.5f)*bin;//distance that corresponds to the current histogram bin
						lI += (Nrij[i] * __sinf(qrij)) / (qrij + 0.000001f);//scattering intensity without form factors
					}
				}
				float lFFij = 2.f * FFi[iq] * FFj[iq];
				I[blockIdx.x * Nq + iq] += lI * lFFij;//multiplying intensity by form-factors and storing the results in global memory
			}
		}
		__syncthreads();//synchronizing threads before the next iteration step
	}
}

//Computes the neutron scattering intensity (powder diffraction pattern) using the histogram of interatomic distances
template <unsigned int Size>__global__ void calcIntHistKernelNeutron(float *I, float SLij, const float *q, unsigned int Nq, const unsigned long long int *rij_hist, unsigned int iBinSt, unsigned int Nhist, unsigned int MaxBinsPerBlock, float bin){
	//see comments in the calcIntHistKernelXray() kernel
	__shared__ long long int Nrij[Size];
	Nrij[threadIdx.x] = 0;
	__syncthreads();
	unsigned int iBegin = iBinSt + blockIdx.x * MaxBinsPerBlock;
	unsigned int iEnd = MIN(Nhist, iBegin + MaxBinsPerBlock);
	if (iEnd < iBegin) return;
	unsigned int Niter = (iEnd - iBegin) / blockDim.x + BOOL((iEnd - iBegin) % blockDim.x);
	unsigned int Nqiter = (Nq / blockDim.x) + BOOL(Nq % blockDim.x);
	for (unsigned int iter = 0; iter < Niter; iter++){
		unsigned int NiterFin = MIN(iEnd - iBegin - iter * blockDim.x, blockDim.x);
		if (threadIdx.x < NiterFin) Nrij[threadIdx.x] = rij_hist[iBegin + iter * blockDim.x + threadIdx.x];
		__syncthreads();
		for (unsigned int iterq = 0; iterq < Nqiter; iterq++) {
			unsigned int iq = iterq * blockDim.x + threadIdx.x;
			if (iq < Nq) {
				float lI = 0, qrij;
				float lq = q[iq];
				for (unsigned int i = 0; i < NiterFin; i++) {
					if (Nrij[i]){
						qrij = lq * ((float)(iBegin + iter * blockDim.x + i) + 0.5f) * bin;
						lI += (Nrij[i] * __sinf(qrij)) / (qrij + 0.000001f);
					}
				}
				I[blockIdx.x * Nq + iq] += 2.f * lI * SLij;
			}
		}
		__syncthreads();
	}
}

//Organazies the computations of the scattering intensity (powder diffraction pattern) using the histogram of interatomic distances with CUDA
void calcInt1DHistCuda(int DeviceNUM, double **I, const unsigned long long int *rij_hist, const unsigned int *NatomEl, const config *cfg, const float * const * dFF, vector<double> SL, const float *dq, unsigned int Ntot){
	unsigned int BlockSize = BlockSize1Dlarge;//setting the size of the thread blocks to 1024 (default)
	float *hI = NULL, *dI = NULL;//host and device arrays for scattering intensity
	*I = new double[cfg->q.N];
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DeviceNUM);//getting device properties
	int cc = deviceProp.major * 10 + deviceProp.minor;//device compute capability
	if (cc < 30) BlockSize = BlockSize1Dmedium;//setting the size of the thread blocks to 512 for the devices with CC < 3.0
	unsigned int GridSize = MIN(256, cfg->Nhist / BlockSize + BOOL(cfg->Nhist % BlockSize));
	unsigned int MaxBinsPerBlock = cfg->Nhist / GridSize + BOOL(cfg->Nhist % GridSize);
	unsigned int GFLOPS = GetGFLOPS(deviceProp);//theoretical peak GPU performance
	if (deviceProp.kernelExecTimeoutEnabled)	{//killswitch is enabled, so the time limit should not be exceeded
		const double tmax = 0.02; //maximum kernel time execution in seconds
		const double k = 1.5e-5; // t = k * Nq * MaxBinsPerBlock / GFLOPS
		MaxBinsPerBlock = MIN((unsigned int)(tmax * GFLOPS / (k * cfg->q.N)), MaxBinsPerBlock);
	}
	unsigned int Isize = GridSize * cfg->q.N;//each block writes to it's own copy of scattering intensity array
	hipMalloc(&dI, Isize * sizeof(float));//allocating the device memory for the scattering intensity array
	unsigned int GSzero = MIN(65535, Isize / BlockSize + BOOL(Isize % BlockSize));//grid size for zero1DFloatArrayKernel
	for (unsigned int iter = 0; iter < Isize / BlockSize + BOOL(Isize % BlockSize); iter += GSzero) zero1DFloatArrayKernel << <GSzero, BlockSize >> >(dI + iter*BlockSize, Isize - iter*BlockSize);//reseting intensity array
	hipDeviceSynchronize();//synchronizing before calculation starts
	unsigned int Nstart = 0, GSadd = cfg->q.N / BlockSize1Dsmall + BOOL(cfg->q.N % BlockSize1Dsmall);//grid size for addIKernelXray/addIKernelNeutron
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {
		if (cfg->source == xray) addIKernelXray << <GSadd, BlockSize1Dsmall >> > (dI, dFF[iEl], cfg->q.N, NatomEl[iEl]);//add contribution form diagonal (i==j) elements in Debye sum
		else addIKernelNeutron << <GSadd, BlockSize1Dsmall >> > (dI, cfg->q.N, float(SQR(SL[iEl]) * NatomEl[iEl]));
		hipDeviceSynchronize();//synchronizing before main calculation starts
		for (unsigned int jEl = iEl; jEl < cfg->Nel; jEl++, Nstart += cfg->Nhist){
			for (unsigned int iBin = 0; iBin < cfg->Nhist; iBin += GridSize * MaxBinsPerBlock) {//iterations to avoid killswitch triggering
				/*float time;
				hipEvent_t start, stop;
				hipEventCreate(&start);
				hipEventCreate(&stop);
				hipEventRecord(start, 0);*/
				if (cfg->source == xray) {//Xray
					if (cc >= 30) calcIntHistKernelXray <BlockSize1Dlarge> << <GridSize, BlockSize >> > (dI, dFF[iEl], dFF[jEl], dq, cfg->q.N, rij_hist + Nstart, iBin, cfg->Nhist, MaxBinsPerBlock, float(cfg->hist_bin));
					else calcIntHistKernelXray <BlockSize1Dmedium> << <GridSize, BlockSize >> > (dI, dFF[iEl], dFF[jEl], dq, cfg->q.N, rij_hist + Nstart, iBin, cfg->Nhist, MaxBinsPerBlock, float(cfg->hist_bin));
				}
				else {//neutron
					if (cc >= 30) calcIntHistKernelNeutron <BlockSize1Dlarge> << <GridSize, BlockSize >> > (dI, float(SL[iEl] * SL[jEl]), dq, cfg->q.N, rij_hist + Nstart, iBin, cfg->Nhist, MaxBinsPerBlock, float(cfg->hist_bin));
					else calcIntHistKernelNeutron <BlockSize1Dmedium> << <GridSize, BlockSize >> > (dI, float(SL[iEl] * SL[jEl]), dq, cfg->q.N, rij_hist + Nstart, iBin, cfg->Nhist, MaxBinsPerBlock, float(cfg->hist_bin));
				}
				/*hipEventRecord(stop, 0);
				hipEventSynchronize(stop);
				hipEventElapsedTime(&time, start, stop);
				cout << "calcIntHistKernel execution time is: " << time << " ms\n" << endl;*/
				hipDeviceSynchronize();//synchronizing before the next iteration step
			}
		}
	}
	sumIKernel << <GSadd, BlockSize1Dsmall >> >(dI, cfg->q.N, GridSize);//summing intensity copies
	hipDeviceSynchronize();//synchronizing threads before multiplying the intensity by a polarization factor
	if (cfg->PolarFactor) PolarFactor1DKernel << <GSadd, BlockSize1Dsmall >> >(dI, cfg->q.N, dq, float(cfg->lambda));
	hI = new float[cfg->q.N];
	hipMemcpy(hI, dI, cfg->q.N * sizeof(float), hipMemcpyDeviceToHost);//copying intensity array from the device to the host
	hipFree(dI);//deallocating memory for intensity array
	for (unsigned int iq = 0; iq < cfg->q.N; iq++) (*I)[iq] = double(hI[iq]) / Ntot;//normalizing
	delete[] hI;
}

//Computes the partial radial distribution function (RDF)
__global__ void calcPartialRDFkernel(float *dPDF, const unsigned long long int *rij_hist, unsigned int Nhist, float mult) {
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < Nhist) dPDF[i] = rij_hist[i] * mult;
}

//Computes the partial pair distribution function (PDF)
__global__ void calcPartialPDFkernel(float *dPDF, const unsigned long long int *rij_hist, unsigned int Nhist, float mult, float bin) {
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < Nhist) {
		float r = (i + 0.5f) * bin;
		dPDF[i] = rij_hist[i] * (mult / SQR(r));
	}
}

//Computes the partial reduced pair distribution function(rPDF)
__global__ void calcPartialRPDFkernel(float *dPDF,const unsigned long long int *rij_hist, unsigned int Nhist, float mult, float submult, float bin) {
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < Nhist) {
		float r = (i + 0.5f) * bin;
		dPDF[i] = rij_hist[i] * (mult / r) - submult * r;
	}
}

//Computes the total PDF using the partial PDFs
__global__ void calcPDFkernel (float *dPDF,unsigned int Nstart,unsigned int Nhist,float multIJ) {
	unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < Nhist) 	dPDF[i] += dPDF[Nstart + i] * multIJ;
}

//Depending on the computational scenario organazies the computations of the scattering intensity (powder diffraction pattern) or PDF using the histogram of interatomic distances with CUDA
void calcPDFandDebyeCuda(int DeviceNUM, double **I, double **PDF, const config *cfg, const unsigned int *NatomEl, const float4 *ra, const float * const * dFF, vector<double> SL, const float *dq) {
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	unsigned long long int *rij_hist = NULL;//array for pair-distribution histogram (device only)
	calcHistCuda(DeviceNUM, &rij_hist, ra, NatomEl, cfg->Nel, cfg->Nhist, float(cfg->hist_bin));//calculating the histogram
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "Histogram calculation time: " << time / 1000 << " s" << endl;
	unsigned int Ntot = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) Ntot += NatomEl[iEl];//calculating the total number of atoms
	if (cfg->scenario > Debye_hist) {//calculating the PDFs
		hipEventRecord(start, 0);
		unsigned int BlockSize = BlockSize1Dmedium;
		unsigned int NPDF = (1 + (cfg->Nel * (cfg->Nel + 1)) / 2) * cfg->Nhist, NPDFh = NPDF;//total PDF array size (full (cfg->Nhist) + partial (cfg->Nhist*(cfg->Nel*(cfg->Nel + 1)) / 2) )
		if (!cfg->PrintPartialPDF) NPDFh = cfg->Nhist;//if the partial PDFs are not needed, we are not copying them to the host
		*PDF = new double[NPDFh];//resulting array of doubles for PDF
		float *hPDF = NULL, *dPDF = NULL;
		hipMalloc(&dPDF, NPDF * sizeof(float));//allocating the device memory for PDF array
		float Faverage2 = 0;
		for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {
			Faverage2 += float(SL[iEl] * NatomEl[iEl]); //calculating the average form-factor
		}
		Faverage2 /= Ntot;
		Faverage2 *= Faverage2;//and squaring it
		//the size of the histogram array may exceed the maximum number of thread blocks in the grid (65535 for the devices with CC < 3.0) multiplied by the thread block size (512 for devices with CC < 2.0 or 1024 for others)
		//so any operations on histogram array should be performed iteratively
		unsigned int GSzero = MIN(65535, NPDF / BlockSize + BOOL(NPDF % BlockSize));//grid size for zero1DFloatArrayKernel
		for (unsigned int iter = 0; iter < NPDF / BlockSize + BOOL(NPDF % BlockSize); iter += GSzero)	zero1DFloatArrayKernel << <NPDF / BlockSize + BOOL(NPDF % BlockSize), BlockSize >> >(dPDF + iter*BlockSize, NPDF - iter*BlockSize);//reseting the PDF array
		hipDeviceSynchronize();//synchronizing before calculation starts
		unsigned int Nstart = 0, GridSizeMax = (cfg->Nhist - 1) / BlockSize + BOOL((cfg->Nhist - 1) % BlockSize), GridSize = MIN(65535, GridSizeMax);//grid size for main kernels
		for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {
			for (unsigned int jEl = iEl; jEl < cfg->Nel; jEl++, Nstart += cfg->Nhist){
				float mult ,sub;
				switch (cfg->PDFtype){
					case typeRDF://calculating partial RDFs
						mult = 2.f / (float(cfg->hist_bin) * Ntot);
						for (unsigned int iter = 0; iter < GridSizeMax; iter += GridSize)	calcPartialRDFkernel << <GridSize, BlockSize >> > (dPDF + iter*BlockSize + cfg->Nhist + Nstart, rij_hist + iter*BlockSize + Nstart, cfg->Nhist - iter*BlockSize, mult);
						break;
					case typePDF://calculating partial PDFs
						mult = 0.5f / (PIf*float(cfg->hist_bin*cfg->p0)*Ntot);
						for (unsigned int iter = 0; iter < GridSizeMax; iter += GridSize) calcPartialPDFkernel << <GridSize, BlockSize >> > (dPDF + iter*BlockSize + cfg->Nhist + Nstart, rij_hist + iter*BlockSize + Nstart, cfg->Nhist - iter*BlockSize, mult, float(cfg->hist_bin));
						break;
					case typeRPDF://calculating partial rPDFs
						mult = 2.f / (float(cfg->hist_bin) * Ntot);
						(jEl > iEl) ? sub = 8.f * PIf * float(cfg->p0) * float(NatomEl[iEl]) * float(NatomEl[jEl]) / SQR(float(Ntot)) : sub=4.f * PIf * float(cfg->p0) * SQR(float(NatomEl[iEl])) / SQR(float(Ntot));
						for (unsigned int iter = 0; iter < GridSizeMax; iter += GridSize) calcPartialRPDFkernel << <GridSize, BlockSize >> > (dPDF + iter * BlockSize + cfg->Nhist + Nstart, rij_hist + iter * BlockSize + Nstart, cfg->Nhist - iter * BlockSize, mult, sub, float(cfg->hist_bin));
						break;
				}
			}
		}
		hipDeviceSynchronize();//synchronizing before calculating the full PDF
		Nstart = cfg->Nhist;
		for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {//calculating full PDF by summing partial PDFs
			for (unsigned int jEl = iEl; jEl < cfg->Nel; jEl++, Nstart += cfg->Nhist){
				float multIJ = float(SL[iEl] * SL[jEl]) / Faverage2;
				for (unsigned int iter = 0; iter < GridSizeMax; iter += GridSize) calcPDFkernel << <GridSize, BlockSize >> > (dPDF + iter*BlockSize, Nstart, cfg->Nhist - iter*BlockSize, multIJ);
				hipDeviceSynchronize();//synchronizing before adding next partial PDF to the full PDF
			}
		}
		hPDF = new float[NPDFh];
		hipMemcpy(hPDF, dPDF, NPDFh * sizeof(float), hipMemcpyDeviceToHost);//copying the PDF from the device to the host
		for (unsigned int i = 0; i < NPDFh; i++) (*PDF)[i] = double(hPDF[i]);//converting into double
		delete[] hPDF;
		if (dPDF != NULL) hipFree(dPDF);
		hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
	    hipEventElapsedTime(&time, start, stop);
	    cout << "PDF calculation time: " << time/1000 << " s" << endl;
	}
	if ((cfg->scenario == Debye_hist) || (cfg->scenario == DebyePDF)) {
		hipEventRecord(start, 0);
		calcInt1DHistCuda(DeviceNUM, I, rij_hist, NatomEl, cfg, dFF, SL, dq, Ntot);//calculating the scattering intensity using the pair-distribution histogram
		hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
	    hipEventElapsedTime(&time, start, stop);
	    cout << "1D pattern calculation time: " << time / 1000 << " s" << endl;
	}
	if (rij_hist != NULL) hipFree(rij_hist);//deallocating memory for pair distribution histogram
}

//Computes the neutron scattering intensity (powder diffraction pattern) using the histogram of interatomic distances
template <unsigned int BlockSize2D> __global__ void calcIntDebyeKernelXray(float *I, const float *FFi, const float *FFj, const float *q, unsigned int Nq, const float4 *ri, const float4 *rj, unsigned int iMax, unsigned int jMax, bool diag){
	if ((diag) && (blockIdx.x < blockIdx.y)) return; //we need to calculate inter-atomic distances only for j > i, so if we are in the diagonal grid, all the subdiagonal blocks (for which j < i for all threads) do nothing and return
	unsigned int jt = threadIdx.x, it = threadIdx.y;
	unsigned int j = blockIdx.x * BlockSize2D + jt;
	unsigned int iCopy = blockIdx.y * BlockSize2D + jt; //jt!!! memory transaction are performed by the threads of the same warp to coalesce them
	unsigned int i = blockIdx.y * BlockSize2D + it;
	__shared__ float3 ris[BlockSize2D], rjs[BlockSize2D]; //cache arrays for the atomic coordinates (we use float3 here to avoid bank conflicts)
	__shared__ float rij[BlockSize2D][BlockSize2D]; //cache array for inter-atomic distances
	rij[it][jt] = 0; //reseting inter-atomic distances array
	if ((it == 0) && (j < jMax)) { //copying the atomic coordinates for j-th (column) atoms (only the threads of the first warp (half-warp for CC < 2.0) are used)
		float4 rt = rj[j]; //we cannot copy float4 to float3 directly (without breaking the transaction coalescing) so the temporary variable in local memory is used
		rjs[jt] = make_float3(rt.x, rt.y, rt.z); //and now converting to float3
	}
	if ((it == 2) && (iCopy < iMax)) { //the same for i-th (row) atoms (only the threads of the third warp (first half-warp of the second warp for CC < 2.0) are used)
		float4 rt = ri[iCopy];
		ris[jt] = make_float3(rt.x, rt.y, rt.z);
	}
	__syncthreads(); //synchronizing threads to ensure that the copying is complete
	if (!diag){
		if ((j < jMax) && (i < iMax)) rij[it][jt] = length(ris[it] - rjs[jt]);//calculating distances
	}
	else{//we are in diagonal grid
		if ((j < jMax) && (i < iMax) && (j > i)) rij[it][jt] = length(ris[it] - rjs[jt]);//all the subdiagonal blocks already quit, but we have diagonal blocks (blockIdx.x == blockIdx.y), so we should check if j > i
	}
	__syncthreads();//synchronizing threads to ensure that the calculation of the distances is complete
	iMax = MIN(BlockSize2D, iMax - blockIdx.y * BlockSize2D); //last i-th (row) atom index for the current block
	jMax = MIN(BlockSize2D, jMax - blockIdx.x * BlockSize2D); //last j-th (column) atom index for the current block
	for (unsigned int iterq = 0; iterq < Nq; iterq += SQR(BlockSize2D)) {//if Nq > SQR(BlockSize2D) there will be threads that compute more than one element of the intensity array
		unsigned int iq = iterq + it * BlockSize2D + jt;
		if (iq < Nq) {//checking for array margin
			float lI = 0, qrij;
			float lq = q[iq];//copying the scattering vector magnitude to the local memory
			if ((diag) && (blockIdx.x == blockIdx.y)) {//diagonal blocks, j starts from i + 1
				for (i = 0; i < iMax; i++) {
#pragma unroll 8//unrolling to speed up the performance
					for (j = i + 1; j < jMax; j++) {
						qrij = lq * rij[i][j];
						lI += __sinf(qrij) / (qrij + 0.000001f); //scattering intensity without form-factors
					}
				}
			}
			else {//j starts from 0
				for (i = 0; i < iMax; i++) {
#pragma unroll 8
					for (j = 0; j < jMax; j++) {
						qrij = lq * rij[i][j];
						lI += __sinf(qrij) / (qrij + 0.000001f);
					}
				}
			}
			I[Nq * (gridDim.x * blockIdx.y + blockIdx.x) + iq] += 2.f * lI * FFi[iq] * FFj[iq]; //multiplying the intensity by form-factors and storing the results in the global memory (2.f is for j < i part)
		}
	}
}

//Computes the neutron scattering intensity (powder diffraction pattern) using the original Debye equation (without the histogram approximation)
template <unsigned int BlockSize2D> __global__ void calcIntDebyeKernelNeutron(float *I, float SLij, const float *q, unsigned int Nq, const float4 *ri, const float4 *rj, unsigned int iMax, unsigned int jMax, bool diag){
	//see comments in the calcIntDebyeKernelXray() kernel
	if ((diag) && (blockIdx.x < blockIdx.y)) return;
	unsigned int jt = threadIdx.x, it = threadIdx.y;
	unsigned int j = blockIdx.x * BlockSize2D + jt;
	unsigned int iCopy = blockIdx.y * BlockSize2D + jt; //jt!!!
	unsigned int i = blockIdx.y * BlockSize2D + it;
	__shared__ float3 ris[BlockSize2D], rjs[BlockSize2D];
	__shared__ float rij[BlockSize2D][BlockSize2D];
	rij[it][jt] = 0;
	if ((it == 0) && (j < jMax)) {
		float4 rt = rj[j];
		rjs[jt] = make_float3(rt.x, rt.y, rt.z);
	}
	if ((it == 2) && (iCopy < iMax)) {
		float4 rt = ri[iCopy];
		ris[jt] = make_float3(rt.x, rt.y, rt.z);
	}
	__syncthreads();
	if (!diag){
		if ((j < jMax) && (i < iMax)) rij[it][jt] = length(ris[it] - rjs[jt]);
	}
	else{
		if ((j < jMax) && (i < iMax) && (j > i)) rij[it][jt] = length(ris[it] - rjs[jt]);
	}
	__syncthreads();
	iMax = MIN(BlockSize2D, iMax - blockIdx.y * BlockSize2D);
	jMax = MIN(BlockSize2D, jMax - blockIdx.x * BlockSize2D);
	for (unsigned int iterq = 0; iterq < Nq; iterq += SQR(BlockSize2D)) {
		unsigned int iq = iterq + it * BlockSize2D + jt;
		if (iq < Nq) {
			float lI = 0, qrij;
			float lq = q[iq];
			if ((diag) && (blockIdx.x == blockIdx.y)) {
				for (i = 0; i < iMax; i++) {
#pragma unroll 8
					for (j = i + 1; j < jMax; j++) {
						qrij = lq * rij[i][j];
						lI += __sinf(qrij) / (qrij + 0.000001f);
					}
				}
			}
			else {
				for (i = 0; i < iMax; i++) {
#pragma unroll 8
					for (j = 0; j < jMax; j++) {
						qrij = lq * rij[i][j];
						lI += __sinf(qrij) / (qrij + 0.000001f);
					}
				}
			}
			I[Nq * (gridDim.x * blockIdx.y + blockIdx.x) + iq] += 2.f * lI * SLij;
		}
	}
}

//Organazies the computations of the scattering intensity(powder diffraction pattern) using the original Debye equation(without the histogram approximation) with CUDA
void calcIntDebyeCuda(int DeviceNUM, double **I, const config *cfg, const unsigned int *NatomEl, const float4 *ra, const float * const * dFF, vector<double> SL, const float *dq){
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	unsigned int BlockSize2D = BlockSize2Dsmall;//setting block size to 32x32 (default)
	float *dI = NULL, *hI = NULL; //host and device arrays for scattering intensity
	unsigned int Ntot = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) Ntot += NatomEl[iEl]; //calculating total number of atoms
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DeviceNUM);//getting the device properties
	size_t free, total;
	hipMemGetInfo(&free, &total);//checking the amount of free GPU memory	
	unsigned int GridSizeExecMax = MIN(128, (unsigned int)(sqrtf(0.5f * free / (cfg->q.N * sizeof(float)))));//we use two-dimensional grid here, so checking the amount of free memory is really important 
	unsigned int BlockSize = SQR(BlockSize2D);//total number of threads per block
	unsigned int GFLOPS = GetGFLOPS(deviceProp); //theoretical peak GPU performance
	if (deviceProp.kernelExecTimeoutEnabled)	{//killswitch is enabled, so the time limit should not be exceeded
		const double tmax = 0.02; //maximum kernel time execution in seconds
		const double k = 5.e-8; // t = k * GridSizeExecMax^2 * BlockSize2D^2 * cfg->q.N / GFLOPS
		GridSizeExecMax = MIN((unsigned int)(sqrt(tmax * GFLOPS / (k * cfg->q.N)) / BlockSize2D), GridSizeExecMax);
	}
	unsigned int Isize = SQR(GridSizeExecMax) * cfg->q.N;//total size of the intensity array
	hipError_t err=hipMalloc(&dI, Isize * sizeof(float));//allocating memory for the intensity array and checking for errors
	if (err != hipSuccess) cout << "Error in calcIntDebyeCuda(), hipMalloc(dI): " << hipGetErrorString(err) << endl;
	unsigned int GSzero = MIN(65535, Isize / BlockSize + BOOL(Isize % BlockSize));//grid size for zero1DFloatArrayKernel
	for (unsigned int iter = 0; iter < Isize / BlockSize + BOOL(Isize % BlockSize); iter += GSzero) zero1DFloatArrayKernel << <GSzero, BlockSize >> >(dI + iter*BlockSize, Isize - iter*BlockSize);//reseting the intensity array
	hipDeviceSynchronize();//synchronizing before calculation starts
	dim3 blockgrid(BlockSize2D, BlockSize2D);
	unsigned int iAtomST = 0, jAtom0, GSadd = cfg->q.N / BlockSize1Dsmall + BOOL(cfg->q.N % BlockSize1Dsmall);//grid size for addIKernelXray/addIKernelNeutron
	bool diag = false;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iAtomST += NatomEl[iEl], iEl++) {
		if (cfg->source == xray) addIKernelXray << <GSadd, BlockSize1Dsmall >> > (dI, dFF[iEl], cfg->q.N, NatomEl[iEl]);//adding contribution from diagonal (i==j) elements in Debye sum
		else addIKernelNeutron << <GSadd, BlockSize1Dsmall >> > (dI, cfg->q.N, float(SQR(SL[iEl]) * NatomEl[iEl]));
		hipDeviceSynchronize();//synchronizing before main calculation starts
		unsigned int jAtomST = iAtomST;
		for (unsigned int jEl = iEl; jEl < cfg->Nel; jAtomST += NatomEl[jEl], jEl++) {
			for (unsigned int iAtom = 0; iAtom < NatomEl[iEl]; iAtom += BlockSize2D*GridSizeExecMax){
				unsigned int GridSizeExecY = MIN((NatomEl[iEl] - iAtom) / BlockSize2D + BOOL((NatomEl[iEl] - iAtom) % BlockSize2D), GridSizeExecMax);//Y-size of grid on current step
				unsigned int iMax = MIN(BlockSize2D * GridSizeExecY, NatomEl[iEl] - iAtom);//last i-th (row) atom in current grid
				(iEl == jEl) ? jAtom0 = iAtom : jAtom0 = 0;
				for (unsigned int jAtom = jAtom0; jAtom < NatomEl[jEl]; jAtom += BlockSize2D*GridSizeExecMax){
					unsigned int GridSizeExecX = MIN((NatomEl[jEl] - jAtom) / BlockSize2D + BOOL((NatomEl[jEl] - jAtom) % BlockSize2D), GridSizeExecMax);//X-size of grid on current step
					unsigned int jMax = MIN(BlockSize2D * GridSizeExecX, NatomEl[jEl] - jAtom);//last j-th (column) atom in current grid
					dim3 grid(GridSizeExecX, GridSizeExecY);
					(iAtomST + iAtom == jAtomST + jAtom) ? diag = true : diag = false;//checking if we are in diagonal grid
					/*float time;
					hipEvent_t start, stop;
					hipEventCreate(&start);
					hipEventCreate(&stop);
					hipEventRecord(start, 0);*/
					if (cfg->source == xray) {
						calcIntDebyeKernelXray <BlockSize2Dsmall> << <grid, blockgrid >> > (dI, dFF[iEl], dFF[jEl], dq, cfg->q.N, ra + iAtomST + iAtom, ra + jAtomST + jAtom, iMax, jMax, diag);
					}
					else {//neutron
						calcIntDebyeKernelNeutron <BlockSize2Dsmall> << <grid, blockgrid >> > (dI, float(SL[iEl] * SL[jEl]), dq, cfg->q.N, ra + iAtomST + iAtom, ra + jAtomST + jAtom, iMax, jMax, diag);
					}
					hipDeviceSynchronize();//synchronizing before launching next kernel (it will write the data to the same array)
					/*hipEventRecord(stop, 0);
					hipEventSynchronize(stop);
					hipEventElapsedTime(&time, start, stop);
					cout << "calcIntDebyeKernel execution time is: " << time << " ms\n" << endl;*/
				}
			}
		}
	}
	sumIKernel << <GSadd, BlockSize1Dsmall >> >(dI, cfg->q.N, SQR(GridSizeExecMax));//summing intensity copies
	hipDeviceSynchronize();//synchronizing before multiplying intensity by a polarization factor
	if (cfg->PolarFactor) PolarFactor1DKernel << <GSadd, BlockSize1Dsmall >> >(dI, cfg->q.N, dq, float(cfg->lambda));
	hI = new float [cfg->q.N];
	*I = new double[cfg->q.N];
	hipMemcpy(hI, dI, cfg->q.N * sizeof(float), hipMemcpyDeviceToHost);//copying the resulting scattering intensity from the device to the host
	for (unsigned int iq = 0; iq < cfg->q.N; iq++) (*I)[iq] = double(hI[iq]) / Ntot;//normalizing
	hipFree(dI);//deallocating device memory for intensity array
	delete[] hI;
	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "1D pattern calculation time: " << time / 1000 << " s" << endl;
}

//Computes the partial scattering intensity (*Ipart) from the partials sums (*I) computed by different thread blocks
__global__ void sumIpartialKernel(float *I, float *Ipart, unsigned int Nq, unsigned int Nsum){
	unsigned int iq = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int ipart = blockIdx.y * Nsum * Nq;
	if (iq < Nq) {
		for (unsigned int j = 1; j < Nsum; j++)	I[ipart + iq] += I[ipart + j * Nq + iq];
		Ipart[(blockIdx.y + 1) * Nq + iq] = I[ipart + iq];
	}
}

//Computes the total scattering intensity (powder diffraction pattern) using the partial scattering intensity
__global__ void integrateIpartialKernel(float *I, unsigned int Nq, unsigned int Nparts){
	unsigned int iq = blockDim.x * blockIdx.x + threadIdx.x;
	if (iq<Nq) {
		I[iq] = 0;
		for (unsigned int ipart = 1; ipart < Nparts + 1; ipart++)	I[iq] += I[ipart * Nq + iq];
	}
}

//Organazies the computations of the scattering intensity (powder diffraction pattern) using the original Debye equation (without the histogram approximation) with CUDA
void calcIntPartialDebyeCuda(int DeviceNUM, double **I, const config *cfg, const unsigned int *NatomEl, const float4 *ra, const float * const * dFF, vector<double> SL, const float *dq, const block *Block){
	float time;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	unsigned int GridSizeExecMax, BlockSize2D = BlockSize2Dsmall, BlockSize, Nparts = (cfg->Nblocks * (cfg->Nblocks + 1)) / 2;
	float *dI = NULL, *dIpart = NULL, *hI = NULL;
	unsigned int Ntot = 0, *NatomElBlock;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) Ntot += NatomEl[iEl];
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, DeviceNUM);
	BlockSize = SQR(BlockSize2D);
	unsigned int GFLOPS = GetGFLOPS(deviceProp); //theoretical peak GPU performance
	size_t free, total;
	hipMemGetInfo(&free, &total);
	GridSizeExecMax = MIN(128, (unsigned int)(sqrtf(0.5f * free / (Nparts * cfg->q.N * sizeof(float)))));
	if (deviceProp.kernelExecTimeoutEnabled)	{
		//killswitch enabled, so the time limit should not be exceeded
		const double tmax = 0.02; //maximum kernel time execution in seconds
		const double k = 5.e-8; // t = k * GridSizeExecMax^2 * BlockSize2D^2 / GFLOPS
		GridSizeExecMax = MIN((unsigned int)(sqrt(tmax * GFLOPS / (k * cfg->q.N)) / BlockSize2D), GridSizeExecMax);
	}
	unsigned int IsizeBlock = SQR(GridSizeExecMax) * cfg->q.N, Isize = Nparts * IsizeBlock;//each block writes to it's own copy of scattering intensity
	hipError_t err = hipMalloc(&dI, Isize * sizeof(float));
	if (err != hipSuccess) cout << "Error in calcIntPartialDebyeCuda(), hipMalloc(dI): " << hipGetErrorString(err) << endl;
	unsigned int GSzero = MIN(65535, Isize / BlockSize + BOOL(Isize % BlockSize));
	for (unsigned int iter = 0; iter < Isize / BlockSize + BOOL(Isize % BlockSize); iter += GSzero) zero1DFloatArrayKernel << <GSzero, BlockSize >> >(dI+iter * BlockSize, Isize - iter * BlockSize);
	hipDeviceSynchronize();
	dim3 blockgrid(BlockSize2D, BlockSize2D);
	unsigned int iAtomST = 0, GSadd = cfg->q.N / BlockSize1Dsmall + BOOL(cfg->q.N % BlockSize1Dsmall), Istart;
	bool diag = false;
	NatomElBlock = new unsigned int[cfg->Nel * cfg->Nblocks];
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++) {
		for (unsigned int iB = 0; iB < cfg->Nblocks; iB++){
			NatomElBlock[iEl * cfg->Nblocks + iB] = 0;
			for (unsigned int iBtype = 0; iBtype < Block[iB].Nid; iBtype++) {
				if (Block[iB].id[iBtype] == iEl) {
					NatomElBlock[iEl * cfg->Nblocks + iB] = Block[iB].NatomElAll[iBtype];
					break;
				}
			}
		}
	}
	for (unsigned int iEl = 0; iEl < cfg->Nel; iAtomST += NatomEl[iEl], iEl++) {
		for (unsigned int iB = 0; iB < cfg->Nblocks; iB++){
			Istart = IsizeBlock * (cfg->Nblocks * iB - (iB * (iB + 1)) / 2 + iB);
			if (cfg->source == xray) addIKernelXray << <GSadd, BlockSize1Dsmall >> > (dI + Istart, dFF[iEl], cfg->q.N, NatomElBlock[iEl * cfg->Nblocks + iB]);
			else addIKernelNeutron << <GSadd, BlockSize1Dsmall >> > (dI + Istart, cfg->q.N, float(SQR(SL[iEl]) * NatomElBlock[iEl * cfg->Nblocks + iB]));
		}
		hipDeviceSynchronize();
		unsigned int jAtomST = iAtomST;
		for (unsigned int jEl = iEl; jEl < cfg->Nel; jAtomST += NatomEl[jEl], jEl++) {
			unsigned int iAtomSB = 0;
			for (unsigned int iB = 0; iB < cfg->Nblocks; iAtomSB += NatomElBlock[iEl * cfg->Nblocks + iB], iB++) {
				for (unsigned int iAtom = 0; iAtom < NatomElBlock[iEl * cfg->Nblocks + iB]; iAtom += BlockSize2D*GridSizeExecMax){
					unsigned int GridSizeExecY = MIN((NatomElBlock[iEl * cfg->Nblocks + iB] - iAtom) / BlockSize2D + BOOL((NatomElBlock[iEl * cfg->Nblocks + iB] - iAtom) % BlockSize2D), GridSizeExecMax);
					unsigned int iMax = MIN(BlockSize2D * GridSizeExecY, NatomEl[iEl] - iAtom);
					unsigned int i0 = iAtomST + iAtomSB + iAtom;
					unsigned int jAtomSB = 0;
					for (unsigned int jB = 0; jB < cfg->Nblocks; jAtomSB += NatomElBlock[jEl * cfg->Nblocks + jB], jB++) {
						(jB>iB) ? Istart = IsizeBlock * (cfg->Nblocks * iB - (iB * (iB + 1)) / 2 + jB) : Istart = IsizeBlock * (cfg->Nblocks * jB - (jB * (jB + 1)) / 2 + iB);
						for (unsigned int jAtom = 0; jAtom < NatomElBlock[jEl * cfg->Nblocks + jB]; jAtom += BlockSize2D * GridSizeExecMax){
							unsigned int j0 = jAtomST + jAtomSB + jAtom;
							if (j0 >= i0) {
								unsigned int GridSizeExecX = MIN((NatomElBlock[jEl * cfg->Nblocks + jB] - jAtom) / BlockSize2D + BOOL((NatomElBlock[jEl * cfg->Nblocks + jB] - jAtom) % BlockSize2D), GridSizeExecMax);
								unsigned int jMax = MIN(BlockSize2D * GridSizeExecX, NatomElBlock[jEl * cfg->Nblocks + jB] - jAtom);
								dim3 grid(GridSizeExecX, GridSizeExecY);
								(i0 == j0) ? diag = true : diag = false;
								if (cfg->source == xray) {
									calcIntDebyeKernelXray <BlockSize2Dsmall> << <grid, blockgrid >> > (dI + Istart, dFF[iEl], dFF[jEl], dq, cfg->q.N, ra + i0, ra + j0, iMax, jMax, diag);
								}
								else {
									calcIntDebyeKernelNeutron <BlockSize2Dsmall> << <grid, blockgrid >> > (dI + Istart, float(SL[iEl] * SL[jEl]), dq, cfg->q.N, ra + i0, ra + j0, iMax, jMax, diag);
								}
								hipDeviceSynchronize();
							}
						}
					}					
				}
			}			
		}
	}
	delete[] NatomElBlock;
	unsigned int IpartialSize = (Nparts + 1) * cfg->q.N;
	hipMalloc(&dIpart, IpartialSize*sizeof(float));
	dim3 gridAdd(GSadd, Nparts);
	sumIpartialKernel << <gridAdd, BlockSize1Dsmall >> >(dI, dIpart, cfg->q.N, SQR(GridSizeExecMax));
	hipDeviceSynchronize();
	hipFree(dI);
	integrateIpartialKernel << <GSadd, BlockSize1Dsmall >> > (dIpart, cfg->q.N, Nparts);
	hipDeviceSynchronize();
	dim3 gridPolar(GSadd, Nparts + 1);
	if (cfg->PolarFactor) PolarFactor1DKernel << <gridPolar, BlockSize1Dsmall >> >(dIpart, cfg->q.N, dq, float(cfg->lambda));
	hI = new float[IpartialSize];
	*I = new double[IpartialSize];
	hipMemcpy(hI, dIpart, IpartialSize * sizeof(float), hipMemcpyDeviceToHost);
	for (unsigned int i = 0; i < IpartialSize; i++) (*I)[i] = double(hI[i]) / Ntot;
	hipFree(dIpart);
	delete[] hI;
	hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	cout << "1D pattern calculation time: " << time / 1000 << " s" << endl;
}

//Queries all CUDA devices. Checks and sets the CUDA device number
//Returns 0 if OK and - 1 if no CUDA devices found
int SetDeviceCuda(int *DeviceNUM){
	int nDevices;
	hipGetDeviceCount(&nDevices);
	if (!nDevices) {
		cout << "Error: No CUDA devices found." << endl;
		return -1;
	}
	if (*DeviceNUM > -1){
		if (*DeviceNUM < nDevices){
			hipSetDevice(*DeviceNUM);
			hipDeviceProp_t deviceProp;
			hipGetDeviceProperties(&deviceProp, *DeviceNUM);
			cout << "Selected CUDA device:" << endl;
			GetGFLOPS(deviceProp, true);
			return 0;
		}
		cout << "Error: Unable to set CUDA device " << *DeviceNUM << ". The total number of CUDA devices is " << nDevices << ".\n";
		cout << "Will use the fastest CUDA device." << endl;
	}
	cout << "The following CUDA devices are found.\n";
	hipDeviceProp_t deviceProp;
	unsigned int GFOLPS=0, MaxGFOLPS=0;
	for (int i = 0; i < nDevices; i++) {
		hipGetDeviceProperties(&deviceProp,i);
		cout << "Device " << i << ":" << endl;
		GFOLPS = GetGFLOPS(deviceProp, true);
		if (GFOLPS > MaxGFOLPS) {
			MaxGFOLPS = GFOLPS;
			*DeviceNUM = i;
		}
	}
	cout << "Will use CUDA device " << *DeviceNUM << "." << endl;
	hipSetDevice(*DeviceNUM);
	return 0;
}

//Copies the atomic coordinates (ra), scattering vector magnitude (q) and the x-ray atomic form-factors (FF) to the device memory	
void dataCopyCUDA(const double *q, const config *cfg, const vector < vect3d <double> > *ra, float4 **dra, float ***dFF, float **dq, vector <double*> FF, unsigned int Ntot){
	//copying the main data to the device memory
	if (cfg->scenario != PDFonly) {//we are calculating no only PDFs but the diffraction patterns too
		float *qfloat;//temporary float array for the scattering vector magnitude
		qfloat = new float[cfg->q.N];
		for (unsigned int iq = 0; iq < cfg->q.N; iq++) qfloat[iq] = (float)q[iq];//converting scattering vector magnitude from double to float
		hipMalloc(dq, cfg->q.N * sizeof(float));//allocating memory for the scattering vector magnitude array
		hipMemcpy(*dq, qfloat, cfg->q.N * sizeof(float), hipMemcpyHostToDevice);//copying scattering vector magnitude array from the host to the device
		delete[] qfloat;//deleting temporary array
		if (cfg->source == xray) {
			*dFF = new float*[cfg->Nel];//this array will store pointers to the atomic form-factor arrays stored in the device memory
			for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++){
				(*dFF)[iEl] = NULL;
				float *FFfloat;//temporary float array for the atomic form-factor
				FFfloat = new float[cfg->q.N];
				for (unsigned int iq = 0; iq < cfg->q.N; iq++) FFfloat[iq] = float(FF[iEl][iq]);//converting form-factors from double to float
				hipMalloc(&(*dFF)[iEl], cfg->q.N * sizeof(float));//allocating device memory for the atomic form-factors
				hipMemcpy((*dFF)[iEl], FFfloat, cfg->q.N * sizeof(float), hipMemcpyHostToDevice);//copying form-factors from the host to the device
				delete[] FFfloat;//deleting temporary array
			}
		}
	}
	hipMalloc(dra,Ntot*sizeof(float4));//allocating device memory for the atomic coordinates array
	float4 *hra;//temporary host array for atomic coordinates
	hra=new float4[Ntot];
	unsigned int iAtom = 0;
	for (unsigned int iEl = 0; iEl < cfg->Nel; iEl++){
		for (vector<vect3d <double> >::const_iterator ri = ra[iEl].begin(); ri != ra[iEl].end(); ri++, iAtom++){
			hra[iAtom] = make_float4((float)ri->x, (float)ri->y, (float)ri->z, 0);//converting atomic coordinates from vect3d <double> to float4
		}
	}	
	hipMemcpy(*dra, hra, Ntot * sizeof(float4), hipMemcpyHostToDevice);//copying atomic coordinates from the host to the device
	delete[] hra;//deleting temporary array
}

//Deletes the atomic coordinates (ra), scattering vector magnitude (dq) and the x-ray atomic form-factors (dFF) from the device memory
void delDataFromDevice(float4 *ra,float **dFF,float *dq, unsigned int Nel){
	hipFree(ra);//deallocating device memory for the atomic coordinates array
	if (dq != NULL) hipFree(dq);//deallocating memory for the scattering vector magnitude array
	if (dFF != NULL) {//Xray source
		for (unsigned int i = 0; i < Nel; i++) if (dFF[i] != NULL) hipFree(dFF[i]);//deallocating device memory for the atomic form-factors
		delete[] dFF;//deleting pointer array
	}
	hipDeviceReset();//NVIDIA Profiler works improperly without this
}
#endif
